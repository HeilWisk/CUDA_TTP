#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <vector>
#include <sstream>

// POPULATION CONTROL
#define MAX_COORD 250
#define POPULATION_SIZE 10 //blockPerGrid*blockPerGrid*BLOCK_SIZE*BLOCK_SIZE
#define BLOCK_SIZE 16
#define NUM_EVOLUTIONS 100
#define MUTATION_RATE 0.05
#define ELITISM true
#define TOURNAMENT_SIZE 128
//BLOCKS
//NUM_THREADS

const int blockPerGrid = 8;

#include "headers/item.h"
#include "headers/node.h"
#include "headers/distance.h"
#include "headers/tour.h"
#include "headers/population.h"
#include "headers/gpu_util.h"

#define DIMENSION "DIMENSION:"
#define ITEM_QTY "NUMBER OF ITEMS:"
#define KNAPSACK_CAPACITY "CAPACITY OF KNAPSACK:"
#define MIN_SPEED "MIN SPEED:"
#define MAX_SPEED "MAX SPEED:"
#define RENTING_RATIO "RENTING RATIO:"
#define EDGE_WEIGHT_TYPE "EDGE_WEIGHT_TYPE:"
#define NODE_COORD_SECTION "NODE_COORD_SECTION	(INDEX, X, Y):"
#define ITEMS_SECTION "ITEMS SECTION	(INDEX, PROFIT, WEIGHT, ASSIGNED NODE NUMBER):"

static void HandleError(hipError_t err, const char* file, int line) 
{
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		getchar();
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#pragma region CUDA Kernels

/// <summary>
/// Basic implementation of matrix transpose
/// </summary>
/// <param name="m_dev">- Matrix to be transposed on device memory</param>
/// <param name="t_m_dev">- Matrix Transpose result on device memory</param>
/// <param name="width">- Width of the matrix</param>
/// <param name="height">- Height of the matrix</param>
/// <returns></returns>
__global__ void transpose(node* m_dev, node* t_m_dev, int width, int height) {

	/* Calculate global index for this thread */
	unsigned int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

	/* Copy m_dev[rowIdx][colIdx] to t_m_dev[rowIdx][colIdx] */
	if (colIdx < width && rowIdx < height)
	{
		unsigned int index_in = colIdx + width * rowIdx;
		unsigned int index_out = rowIdx + height * colIdx;
		t_m_dev[index_out] = m_dev[index_in];
		if (t_m_dev[index_out].item_qty > 0)
		{
			t_m_dev[index_out].items = m_dev[index_in].items;
		}
		
	}
}

/// <summary>
/// Optimized Kernel to ensure all global reads and writes are coalesced and to avoid bank conflicts in
/// shared memory. This Kernel is up to 11x faster than "matrix_transpose" kernel.
/// </summary>
/// <param name="m_dev">- Matrix to be transposed on device memory</param>
/// <param name="t_m_dev">- Matrix Transpose result on device memory</param>
/// <param name="width">- Width of the matrix</param>
/// <param name="height">- Height of the matrix</param>
/// <returns></returns>
__global__ void matrixTransposeCoalesced(node* m_dev, node* t_m_dev, int width, int height) {

	__shared__ node block[BLOCK_SIZE][BLOCK_SIZE + 1];

	// Read matrix tile into shared memory
	// Load one element per thread from device memory (m_dev) and store it in transposed order in block[][]
	unsigned int colIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	unsigned int rowIdx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	if ((colIdx < width) && (rowIdx < height))
	{
		unsigned int index_in = rowIdx * width + colIdx;
		block[threadIdx.y][threadIdx.x] = m_dev[index_in];
	}

	// Synchronise to ensure allwrites to block[][] have completed
	__syncthreads();

	// Write the transposed matrix tile to global memory (t_m_dev) in linear order
	colIdx = blockIdx.y * BLOCK_SIZE + threadIdx.x;
	rowIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;
	if ((colIdx < height) && (rowIdx < width))
	{
		unsigned int index_out = rowIdx * height + colIdx;
		t_m_dev[index_out] = block[threadIdx.x][threadIdx.y];
	}
}

/// <summary>
/// Kernel to calculate distances between point matrixes
/// </summary>
/// <param name="m_src_dev">- Matrix with source coodinates</param>
/// <param name="m_dst_dev">- Matrix with destination coordinates</param>
/// <param name="m_dist_dev">- Result Matrix with euclidean distances</param>
/// <param name="m_dist_dev_rows">- Result matrix row count</param>
/// <param name="m_dist_dev_cols">- Result matrix column count</param>
/// <returns></returns>
__global__ void matrixDistances(node* m_src_dev, node* m_dst_dev, distance* m_dist_dev, int m_dist_dev_rows, int m_dist_dev_cols) {

	// Define variables
	const unsigned int width = 1;

	// Calculate global indexes
	unsigned int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

	// Check boundary conditions
	if (rowIdx < m_dist_dev_rows && colIdx < m_dist_dev_cols)
	{
		// Execute distance calculation
		float value = 0;
		int sourceId = 0;
		int destinyId = 0;
		for (int k = 0; k < width; k++)
		{
			sourceId = m_src_dev[rowIdx * width + k].id;
			destinyId = m_dst_dev[k * m_dist_dev_cols + colIdx].id;
			value += pow(m_dst_dev[k * m_dist_dev_cols + colIdx].x - m_src_dev[rowIdx * width + k].x, 2) + pow(m_dst_dev[k * m_dist_dev_cols + colIdx].y - m_src_dev[rowIdx * width + k].y, 2);
		}
		m_dist_dev[rowIdx * m_dist_dev_cols + colIdx].source = sourceId;
		m_dist_dev[rowIdx * m_dist_dev_cols + colIdx].destiny = destinyId;
		m_dist_dev[rowIdx * m_dist_dev_cols + colIdx].value = sqrt(value);
	}
}

/// <summary>
/// 
/// </summary>
/// <param name="state"></param>
/// <param name="seed"></param>
/// <returns></returns>
__global__ void initCuRand(hiprandState* state, time_t seed)
{
	// Calculate global index of the threads for the 2D GRID
	// Global index of every block on the grid
	int block_number_in_grid = blockIdx.x + gridDim.x * blockIdx.y;
	// Global index of every thread in block
	int thread_number_in_block = threadIdx.x + blockDim.x * threadIdx.y;
	// Number of thread per block
	int threads_per_block = blockDim.x * blockDim.y;
	// Global index of every thread on the grid
	int thread_global_index = block_number_in_grid * threads_per_block + thread_number_in_block;

	if (thread_global_index >= POPULATION_SIZE)
		return;

	hiprand_init(seed, thread_global_index, 0, &state[thread_global_index]);
}

/// <summary>
/// 
/// </summary>
/// <param name="population"></param>
/// <param name="distance_table"></param>
/// <param name="node_quantity"></param>
/// <returns></returns>
__global__ void evaluatePopulation(population* population, distance* distance_table, const int node_quantity)
{
	// Get thread ID
	// Global index of every block on the grid
	int block_number_in_grid = blockIdx.x + gridDim.x * blockIdx.y;
	// Global index of every thread in block
	int thread_number_in_block = threadIdx.x + blockDim.x * threadIdx.y;
	// Number of thread per block
	int threads_per_block = blockDim.x * blockDim.y;
	// Global index of every thread on the grid
	int thread_global_index = block_number_in_grid * threads_per_block + thread_number_in_block;

	if (thread_global_index < POPULATION_SIZE)
		evaluateTour(population->tours[thread_global_index], distance_table, node_quantity);
}

/// <summary>
/// 
/// </summary>
/// <param name="population"></param>
/// <param name="randState"></param>
/// <param name="parents"></param>
/// <param name="node_quantity"></param>
/// <param name="item_quantity"></param>
/// <returns></returns>
__global__ void selection(population* population, hiprandState* randState, tour* parents, const int node_quantity, const int item_quantity)
{
	// Get thread global id
	// Global index of every block on the grid
	int block_number_in_grid = blockIdx.x + gridDim.x * blockIdx.y;
	// Global index of every thread in block
	int thread_number_in_block = threadIdx.x + blockDim.x * threadIdx.y;
	// Number of thread per block
	int threads_per_block = blockDim.x * blockDim.y;
	// Global index of every thread on the grid
	int thread_global_index = block_number_in_grid * threads_per_block + thread_number_in_block;

	if (thread_global_index < POPULATION_SIZE)
	{
		parents[thread_global_index * 2] = tournamentSelection(*population, randState, thread_global_index, node_quantity, item_quantity);
		parents[thread_global_index * 2+1] = tournamentSelection(*population, randState, thread_global_index, node_quantity, item_quantity);
	}
}

/// <summary>
/// 
/// </summary>
/// <param name="population"></param>
/// <param name="parents"></param>
/// <param name="random_state"></param>
/// <param name="distance_table"></param>
/// <param name="index"></param>
/// <returns></returns>
__global__ void crossover(population* population, tour* parents, hiprandState* random_state, distance* distance_table, int index)
{
	// Get thread global id
	// Global index of every block on the grid
	int block_number_in_grid = blockIdx.x + gridDim.x * blockIdx.y;
	// Global index of every thread in block
	int thread_number_in_block = threadIdx.x + blockDim.x * threadIdx.y;
	// Number of thread per block
	int threads_per_block = blockDim.x * blockDim.y;
	// Global index of every thread on the grid
	int thread_global_index = block_number_in_grid * threads_per_block + thread_number_in_block;

	if (thread_global_index < POPULATION_SIZE)
	{
		population->tours[thread_global_index].nodes[0] = parents[2 * thread_global_index].nodes[0];
		node node_1 = getValidNextNode(parents[thread_global_index * 2], population->tours[thread_global_index], population->tours[thread_global_index].nodes[index - 1], index);

	}
}

/// <summary>
/// 
/// </summary>
/// <param name="population"></param>
/// <param name="d_state"></param>
/// <returns></returns>
__global__ void mutate(population* population, hiprandState* d_state, const int number_nodes)
{
	// Get thread global id
	// Global index of every block on the grid
	int block_number_in_grid = blockIdx.x + gridDim.x * blockIdx.y;
	// Global index of every thread in block
	int thread_number_in_block = threadIdx.x + blockDim.x * threadIdx.y;
	// Number of thread per block
	int threads_per_block = blockDim.x * blockDim.y;
	// Global index of every thread on the grid
	int thread_global_index = block_number_in_grid * threads_per_block + thread_number_in_block;

	if (thread_global_index < POPULATION_SIZE)
	{
		// Pick random number between 0 and 1
		hiprandState local_state = d_state[thread_global_index];

		// If random number is less than mutation rate, perform mutation (swap to nodes in tour)
		if (hiprand_uniform(&local_state) < MUTATION_RATE)
		{
			int random_number_one = 1 + hiprand_uniform(&local_state) * (number_nodes - 1.0000001);
			int random_number_two = 1 + hiprand_uniform(&local_state) * (number_nodes - 1.0000001);

			node temp = population->tours[thread_global_index].nodes[random_number_one];
			population->tours[thread_global_index].nodes[random_number_one] = population->tours[thread_global_index].nodes[random_number_two];
			population->tours[thread_global_index].nodes[random_number_two] = temp;

			d_state[thread_global_index] = local_state;
		}
	}
}

#pragma endregion

#pragma region CUDA Functions

void cudaCheckError()
{
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
	{
		fprintf(stderr, "CUDA Failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
	}
}

#pragma endregion

/// <summary>
/// Function to find a character position in a string
/// </summary>
/// <param name="stringToSearch">- String to search</param>
/// <param name="characterToFind">- Character to find in the string</param>
/// <returns>Position in the string of the character</returns>
size_t findCharacterPosition(char stringToSearch[], char characterToFind)
{
	size_t stringLength = 0, i, characterPosition = 0;

	stringLength = strlen(stringToSearch);
	for (i = 0; i < stringLength; i++)
	{
		if (stringToSearch[i] == characterToFind)
			characterPosition = i + 1;
	}

	return (characterPosition);
}

/// <summary>
/// Extracts a string from another string
/// </summary>
/// <param name="originalString">- Original string</param>
/// <param name="subString">- Resulting Substring</param>
/// <param name="position">- Initial position where the substring is about to begin</param>
/// <param name="length">- Length of the desired substring</param>
void subString(char originalString[], char subString[], size_t position, size_t length)
{
	int c = 0, d = 0;
	char tempSubString[1000];

	while (c < length) {
		subString[c] = originalString[position + c - 1];
		c++;
	}

	subString[c] = '\0';
	c = 0;

	while (subString[c] != '\0')
	{
		if (subString[c] == ' ' || subString[c] == '	') {
			int temp = c + 1;
			if (subString[temp] != '\0') {
				while ((subString[temp] == ' ' || subString[c] == '	') && subString[temp] != '\0') {
					if (subString[temp] == ' ' || subString[c] == '	')
						c++;
					temp++;
				}
			}
		}
		tempSubString[d] = subString[c];
		c++;
		d++;
	}

	tempSubString[d] = '\0';
	strcpy(subString, tempSubString);
}

/// <summary>
/// Function to count the amount of lines in a file
/// </summary>
/// <param name="fileName">- File path and name of the file to evaluate</param>
/// <returns>Amount of lines in the file</returns>
int countFileLines(char fileName[]) {

	FILE* filePtr;
	int lineCount = 0;
	char chr;

	filePtr = fopen(fileName, "r");
	chr = getc(filePtr);
	while (chr != EOF)
	{
		if (chr == '\n')
			lineCount++;
		chr = getc(filePtr);
	}
	fclose(filePtr);
	return lineCount;
}

/// <summary>
/// Count the rows for a matrix in a file with a given structure
/// </summary>
/// <param name="fileName">- File path and name of the file to evaluate</param>
/// <param name="sectionName">- Section name in the file where the matrix begins</param>
/// <returns>Amount of rows in the matrix</returns>
int countMatrixRows(const char fileName[], const char sectionName[])
{
	FILE* filePtr;
	char str[255], sub[255];
	int lineCount = 0, initialPosition = 0, rows = 0;
	const char openMode[] = "r";

	filePtr = fopen(fileName, openMode);

	while (fgets(str, 100, filePtr) != NULL) {
		if (strncmp(str, sectionName, strlen(sectionName)) == 0) {
			initialPosition = lineCount;
		}
		subString(str, sub, 1, 1);
		if (initialPosition != NULL && lineCount > initialPosition && isdigit(sub[0])) {
			rows++;
		}
		else if (initialPosition != NULL && lineCount > initialPosition && isalpha(sub[0]))
		{
			break;
		}
		lineCount++;
	}
	fclose(filePtr);
	return rows;
}

/// <summary>
/// Validates if a file exits
/// </summary>
/// <param name="path">- File path and name of the file</param>
/// <returns>0: File does not exist, 1: File exist</returns>
int fileExists(const char* path)
{
	// Try to open file
	FILE* fptr = fopen(path, "r");

	// If file doesn't exists
	if (fptr == NULL)
		return 0;

	// File exists hence close file and return true
	fclose(fptr);

	return 1;
}

/// <summary>
/// Extracts matrix from a file with a given structure
/// </summary>
/// <param name="fileName">- File path and name</param>
/// <param name="sectionName">- Section name in the file</param>
/// <param name="rows">- Amount of columns</param>
/// <param name="cols">- Amount of rows</param>
/// <returns>Double pointer matrix of ints</returns>
int** extractMatrixFromFile(const char fileName[], const char sectionName[], int rows, int cols)
{
	FILE* filePtr;
	char str[255], sub[255], * token;
	int lineCount = 0, initialPosition = 0, matrixRow, matrixCol;
	const char openMode[] = "r";

	filePtr = fopen(fileName, openMode);

	// Allocate memory for rows
	int** matrixResult = (int**)malloc(rows * sizeof(int*));
	if (matrixResult == NULL) {
		fprintf(stderr, "Out of Memory");
		exit(0);
	}

	// Allocate memory for columns
	for (int i = 0; i < rows; i++) {
		matrixResult[i] = (int*)malloc(cols * sizeof(int));
		if (matrixResult[i] == NULL) {
			fprintf(stderr, "Out of Memory");
			exit(0);
		}
	}

	while (fgets(str, 100, filePtr) != NULL) {
		if (strncmp(str, sectionName, strlen(sectionName)) == 0) {
			initialPosition = lineCount;
		}
		subString(str, sub, 1, 1);
		if (initialPosition != NULL && lineCount > initialPosition && isdigit(sub[0])) {
			token = strtok(str, "	");
			matrixCol = 0;
			matrixRow = atoi(token) - 1;
			while (token != NULL)
			{
				matrixResult[matrixRow][matrixCol] = atoi(token);
				token = strtok(NULL, "	");
				if (matrixCol < cols)
					matrixCol++;
			}
		}
		else if (initialPosition != NULL && lineCount > initialPosition && isalpha(sub[0]))
		{
			break;
		}
		lineCount++;
	}

	fclose(filePtr);

	return matrixResult;
}

/// <summary>
/// Displays a matrix on screen
/// </summary>
/// <param name="matrix">- Matrix to display</param>
/// <param name="rows">- Amount of rows in the matrix</param>
/// <param name="c">- Amount of columns in the matrix</param>
void display(int** matrix, int rows, int columns) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < columns; j++) {
			printf("%d ", matrix[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

/// <summary>
/// Displays a matrix on screen
/// </summary>
/// <param name="matrix">- Matrix to display</param>
/// <param name="rows">- Amount of rows in the matrix</param>
/// <param name="c">- Amount of columns in the matrix</param>
void display(float** matrix, int rows, int columns) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < columns; j++) {
			printf("%f ", matrix[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

/// <summary>
/// Calculates euclidean distance between a matrix of source points and a matrix of destination points
/// </summary>
/// <param name="srcPoint">- Matrix of source points</param>
/// <param name="dstPoint">- Matrix of destination points</param>
/// <param name="out">- Result matrix with distances</param>
/// <param name="rCount">- Row count</param>
/// <param name="size">- Total size of the result matrix</param>
void euclideanDistanceCPU(node* srcPoint, node* dstPoint, distance* out, int rCount, int size) {
	for (int s = 0; s < size; s++) {
		for (int xSrc = 0; xSrc < rCount; xSrc++) {
			for (int xDst = 0; xDst < rCount; xDst++) {
				out[s].source = srcPoint[xSrc].id;
				out[s].destiny = dstPoint[xDst].id;
				out[s].value = (float)sqrt(pow(dstPoint[xDst].x - srcPoint[xSrc].x, 2) + pow(dstPoint[xDst].y - srcPoint[xSrc].y, 2) * 1.0);
				s++;
			}
		}
	}
}

int main()
{
	/****************************************************************************************************
	* DECLARE VARIABLES
	****************************************************************************************************/

	// File variables
	char file_name[255], str[255], sub[255];
	FILE* fp;
	size_t position;
	const char openMode[] = "r";

	// Problem variables
	int** matrix;
	double knapsack_capacity;
	double minimal_speed;
	double maximun_speed;
	double renting_ratio;
	unsigned int node_quantity;
	unsigned int item_quantity;
	char edge_weight_type[1000];

#pragma region PRINT GPU PROPERTIES
	/****************************************************************************************************
	* PRINT START OF THE PROGRAM
	****************************************************************************************************/
	int count;
	hipDeviceProp_t properties;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	printf("****************************************************************************************\n");
	printf("PROPERTIES OF THE GRAPHICAL PROCESSING UNIT\n");
	printf("****************************************************************************************\n");
	for (int i = 0; i < count; i++)
	{
		HANDLE_ERROR(hipGetDeviceProperties(&properties, i));
		printf("GPU:					%s\n", properties.name);
		printf("Warp Size:				%d\n", properties.warpSize);
		printf("Total Global Memory:			%zd\n", properties.totalGlobalMem);
		printf("Total Constant Memory:			%zd\n", properties.totalConstMem);
		printf("Shared Memory Per Block:		%zd\n", properties.sharedMemPerBlock);
		printf("Multiprocessor:				%d\n", properties.multiProcessorCount);
		printf("Max Threads Per Multiprocessor:		%d\n", properties.maxThreadsPerMultiProcessor);
		printf("Max Blocks Per Multiprocessor:		%d\n", properties.maxBlocksPerMultiProcessor);
		printf("Max Threads Per Block:			%d\n", properties.maxThreadsPerBlock);
	}
	printf("****************************************************************************************\n");
#pragma endregion

#pragma region CAPTURE FILE PATH
	/****************************************************************************************************
	* CAPTURE FILE PATH AND LOAD HIS DATA
	****************************************************************************************************/
	// Ask for the filepath & name where the problem is defined
	printf("Enter name of a file you wish to see\n");
	gets_s(file_name);
	printf("\n");

	// Open the file in read mode
	fp = fopen(file_name, openMode);

	// Validates for errors on file opening
	if (fp == NULL)
	{
		perror("Error while opening the file.\n");
		exit(EXIT_FAILURE);
	}

	// Print headers
	printf("****************************************************************************************\n");
	printf("CONTENTS OF THE FILE:\n");
	printf("****************************************************************************************\n");
	printf("The quantity of lines in the file are:	%d\n", countFileLines(file_name));

	// Obtain general data from file
	while (fgets(str, 100, fp) != NULL) {
		position = findCharacterPosition(str, ':');
		// Extract amount of nodes (cities)
		if (strncmp(str, DIMENSION, strlen(DIMENSION)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			node_quantity = atoi(sub);
			printf("Nodes (Cities):				%d\n", node_quantity);
		}
		// Extract the amount of items
		else if (strncmp(str, ITEM_QTY, strlen(ITEM_QTY)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			item_quantity = atoi(sub);
			printf("Item:					%d\n", item_quantity);
		}
		// Extract the knapsack capacity
		else if (strncmp(str, KNAPSACK_CAPACITY, strlen(KNAPSACK_CAPACITY)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			knapsack_capacity = atof(sub);
			printf("Knapsack Capacity:			%lf\n", knapsack_capacity);
		}
		// Extract the minimal speed
		else if (strncmp(str, MIN_SPEED, strlen(MIN_SPEED)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			minimal_speed = atof(sub);
			printf("Minimum Speed:				%lf\n", minimal_speed);
		}
		// Extract the maximum speed
		else if (strncmp(str, MAX_SPEED, strlen(MAX_SPEED)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			maximun_speed = atof(sub);
			printf("Maximum Speed:				%lf\n", maximun_speed);
		}
		// Extract the renting ratio
		else if (strncmp(str, RENTING_RATIO, strlen(RENTING_RATIO)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			renting_ratio = atof(sub);
			printf("Renting Ratio:				%lf\n", renting_ratio);
		}
		// Extract the edge weight type
		else if (strncmp(str, EDGE_WEIGHT_TYPE, strlen(EDGE_WEIGHT_TYPE)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			strcpy(edge_weight_type, sub);
			printf("Edge Weight Type is			%s", edge_weight_type);
		}
	}

	// Close file
	fclose(fp);
	printf("****************************************************************************************\n");
	printf("\n");
#pragma endregion

	/****************************************************************************************************
	* PRINT CUDA AND GENETIC VALUES
	****************************************************************************************************/
	printf("****************************************************************************************\n");
	printf("PROPERTIES FOR THE PROBLEM\n");
	printf("****************************************************************************************\n");
	printf("THREADS:				PD\n");
	printf("BLOCKS:					PD\n");
	printf("TOURNAMENT SIZE:			PD\n");
	printf("AMOUNT OF EVOLUTIONS:			PD\n");
	printf("****************************************************************************************\n");

#pragma region POPULATION INITIALIZATION CPU
	/*************************************************************************************************
	* POPULATION INITIALIZATION ON HOST (CPU)
	*************************************************************************************************/
	tour initial_tour(node_quantity, item_quantity, false);
	population initial_population;

	// Obtain the items
	// Calculate amount of rows
	unsigned int item_rows = countMatrixRows(file_name, ITEMS_SECTION);
	// Validate file consistency
	if (item_rows != item_quantity)
	{
		perror("The file information is not consistent. Number of items Inconsistency.\n");
		exit(EXIT_FAILURE);
	}
	// Calculate amount of columns
	unsigned int item_columns = 4;
	// Get matrix
	matrix = extractMatrixFromFile(file_name, ITEMS_SECTION, item_quantity, item_columns);
	// Allocate memory for the array of structs
	item* i = (item*)malloc(item_quantity * sizeof(item));
	if (i == NULL) {
		fprintf(stderr, "Out of Memory");
		exit(0);
	}
	// Convert to array of struct
	extractItems(matrix, item_quantity, i);
	// Visualize values for item matrix	
	displayItems(i, item_quantity);
	
	// Obtain nodes
	// Calculate amount of nodes
	unsigned int node_rows = countMatrixRows(file_name, NODE_COORD_SECTION);
	// Validate file consistency
	if (node_rows != node_quantity)
	{
		perror("The file information is not consistent. Number of node Inconsistency.\n");
		exit(EXIT_FAILURE);
	}
	// Calculate amount of columns
	unsigned int node_columns = 3;
	// Get matrix
	matrix = extractMatrixFromFile(file_name, NODE_COORD_SECTION, node_quantity, node_columns);
	// Allocate memory for the array of structs
	node* n = (node*)malloc(node_quantity * sizeof(node));
	if (n == NULL) {
		fprintf(stderr, "Out of Memory");
		exit(0);
	}
	// Convert to array of struct
	extractNodes(matrix, node_quantity, n);

	// Assign items to node
	assignItems(i, item_quantity, n, node_quantity);

	// Print node information
	displayNodes(n, node_quantity);

	// Assign nodes to tour
	defineInitialTour(initial_tour, node_quantity, n);	

	// Calculate distance matrix in CPU
	int distance_matrix_size = node_quantity * node_quantity;
	distance* d = (distance*)malloc(distance_matrix_size * sizeof(distance));
	if (d == NULL) {
		fprintf(stderr, "Out of Memory");
		exit(0);
	}

	euclideanDistanceCPU(n, n, d, node_quantity, distance_matrix_size);
	displayDistance(d, distance_matrix_size);

	// Initialize population by generating POPULATION_SIZE number of
	// permutations of the initial tour, all starting at the same city
	initializePopulationCPU(initial_population, initial_tour, d, POPULATION_SIZE, node_quantity);
	printPopulation(initial_population, POPULATION_SIZE);
#pragma endregion

#pragma region POPULATION INITIALIZATION GPU
	/*************************************************************************************************
	* POPULATION INITIALIZATION ON DEVICE (GPU)
	*************************************************************************************************/

	// Setup execution parameters
	//dim3 grid(node_columns / BLOCK_SIZE, node_rows / BLOCK_SIZE, 1);
	dim3 grid(blockPerGrid, blockPerGrid, 1);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 1);

	// Initialize random values
	hiprandState* d_states;
	HANDLE_ERROR(hipMalloc((void**)&d_states, sizeof(hiprandState) * POPULATION_SIZE * node_quantity));
	initCuRand << <grid, threads >> > (d_states, time(NULL));
	HANDLE_ERROR(hipDeviceSynchronize());

	// 1. hipMalloc a pointer to device memory that hold population
	population* d_initial_population;
	HANDLE_ERROR(hipMalloc((void**)&d_initial_population, sizeof(population)));

	// 2. Create a separate tour pointer on the host.
	tour* d_tour_ptr;
	HANDLE_ERROR(hipMalloc((void**)&d_tour_ptr, sizeof(tour) * POPULATION_SIZE));

	// 3. Create a separate node pointer on the host.
	node* d_node_ptr[POPULATION_SIZE];

	// Allocate memory on device according to population size
	for (int i = 0; i < POPULATION_SIZE; ++i)
	{
		// 4. hipMalloc node storage on the device for node pointer
		HANDLE_ERROR(hipMalloc((void**)&(d_node_ptr[i]), sizeof(node) * node_quantity));
		// 5. hipMemcpy the pointer value of node pointer from host to the device node pointer
		HANDLE_ERROR(hipMemcpy(&(d_tour_ptr[i].nodes), &(d_node_ptr[i]), sizeof(node*), hipMemcpyHostToDevice));
		// Optional: Copy an instantiated object on the host to the device pointer
		HANDLE_ERROR(hipMemcpy(d_node_ptr[i], initial_tour.nodes, sizeof(node) * node_quantity, hipMemcpyHostToDevice));
	}
	// 6. hipMemcpy the pointer value of tour pointer from host to the device population pointer
	HANDLE_ERROR(hipMemcpy(&(d_initial_population->tours), &d_tour_ptr, sizeof(tour*), hipMemcpyHostToDevice));

	/********************************************************************************************************************
	* Calculate Distance Matrix in CUDA
	********************************************************************************************************************/
	// First calculate the matrix transpose
	// Define device pointers
	node* d_node_matrix;
	node* d_node_t_matrix;

	// Allocate memory on device
	HANDLE_ERROR(hipMalloc(&d_node_matrix, node_quantity * sizeof(node)));
	HANDLE_ERROR(hipMalloc(&d_node_t_matrix, node_quantity * sizeof(node)));
	HANDLE_ERROR(hipMemcpy(d_node_matrix, n, node_quantity * sizeof(node), hipMemcpyHostToDevice));	

	// Execute CUDA Matrix Transposition
	printf("Transponiendo la matrix de nodos de tamaño [%d][%d]\n", node_rows, 1);
	transpose << <grid, threads >> > (d_node_matrix, d_node_t_matrix, node_rows, 1);
	HANDLE_ERROR(hipDeviceSynchronize());

	// Copy results from device to host
	node* h_node_t_matrix = (node*)malloc(sizeof(node) * node_quantity);
	HANDLE_ERROR(hipMemcpy(h_node_t_matrix, d_node_t_matrix, sizeof(node) * node_quantity, hipMemcpyDeviceToHost));

	// Show information on screen
	displayNodes(h_node_t_matrix, node_quantity);

	// Calculate size of distance array
	distance* d_distance;
	int distance_size = node_quantity * node_quantity;
	HANDLE_ERROR(hipMalloc(&d_distance, sizeof(distance) * distance_size));
	printf("Calculando la matriz de distancias en GPU\n");
	matrixDistances << <grid, threads >> > (d_node_matrix, d_node_t_matrix, d_distance, node_quantity, node_quantity);
	HANDLE_ERROR(hipDeviceSynchronize());

	//Copy results from device to host
	distance* h_distance = (distance*)malloc(sizeof(distance) * distance_size);
	HANDLE_ERROR(hipMemcpy(h_distance, d_distance, sizeof(distance) * distance_size, hipMemcpyDeviceToHost));

	// Show Data
	displayDistance(h_distance, distance_size);
	
	// Invoke Kernel to generate the initial population on the GPU
	initializePopulationGPU << <grid, threads >> > (d_initial_population, d_distance, node_quantity, item_rows, d_states);
	HANDLE_ERROR(hipDeviceSynchronize());

	//Copy results from device to host
	population h_initial_population;
	HANDLE_ERROR(hipMemcpy(&h_initial_population, d_initial_population, sizeof(population), hipMemcpyDeviceToHost));
	tour* h_tour_ptr = (tour*)malloc(sizeof(tour) * POPULATION_SIZE);
	HANDLE_ERROR(hipMemcpy(h_tour_ptr, d_tour_ptr, sizeof(tour) * POPULATION_SIZE, hipMemcpyDeviceToHost));	
	h_initial_population.tours = h_tour_ptr;
	node* h_node_ptr[POPULATION_SIZE];
	
	for (int p = 0; p < POPULATION_SIZE; ++p)
	{
		h_node_ptr[p] = (node*)malloc(sizeof(node) * node_quantity);
		HANDLE_ERROR(hipMemcpy(h_node_ptr[p], d_node_ptr[p], sizeof(node) * node_quantity, hipMemcpyDeviceToHost));
		h_initial_population.tours[p].nodes = h_node_ptr[p];
	}

	// Print Result
	printPopulation(h_initial_population, POPULATION_SIZE);
#pragma endregion

#pragma region GPU MEMORY ALLOCATION
	/****************************************************************************************************
	* GPU MEMORY ALLOCATION
	****************************************************************************************************/
	//TODO: Evaluar toda la seccion para determinar que se puede quitar y que no, por ahora solo voy a hacer copy-paste
	population* device_population;
	HANDLE_ERROR(hipMalloc((void**)&device_population, sizeof(population)));

	// Array to store parents selected from tournament selection
	tour* device_parents;
	HANDLE_ERROR(hipMalloc((void**)&device_parents, sizeof(tour) * POPULATION_SIZE * 2));

	// Cost table for crossover function (SCX Crossover)
	// TODO: Revisar esta memoria dado que la tabla de costos que se tiene elaborada es con base a estructuras y ya esta generada en GPU
	distance* device_cost_table;
	HANDLE_ERROR(hipMalloc((void**)&device_cost_table, sizeof(distance) * node_quantity * node_quantity));

	// Array for random numbers
	hiprandState* device_state;
	HANDLE_ERROR(hipMalloc((void**)&device_state, POPULATION_SIZE * sizeof(hiprandState)));
	HANDLE_ERROR(hipDeviceSynchronize());

	// Copies data to device for evolution
	HANDLE_ERROR(hipMemcpy(device_population, &h_initial_population, sizeof(population), hipMemcpyHostToDevice));
	// TODO: Revisar con lupa esta linea dado que h_distance esta expresado en otros terminos, especificamente es un arreglo de estructura tipo distancia no flotantes
	HANDLE_ERROR(hipMemcpy(device_cost_table, &h_distance, sizeof(float) * node_quantity * node_quantity, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipDeviceSynchronize());
#pragma endregion

#pragma region TIMED GPU ALGORITHMS
	/****************************************************************************************************
	* TIMED EXECUTION OF EVOLVE POPULATION ON GPU
	****************************************************************************************************/
	float milliseconds;
	hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start));

	HANDLE_ERROR(hipDeviceSynchronize());

	/****************************************************************************************************
	* MAIN LOOP OF TSP
	****************************************************************************************************/
	// Initialize random numbers array for tournament selection	
	initCuRand << <grid, threads >> > (d_states, time(NULL));
	HANDLE_ERROR(hipDeviceSynchronize());

	// Figure out distance and fitness for each individual in population
	evaluatePopulation << <grid, threads >> > (device_population, device_cost_table, node_quantity);
	
	for(int e = 0; e < NUM_EVOLUTIONS; ++e)
	{
		selection << <grid, threads >> > (device_population, device_state, device_parents, node_quantity, item_quantity);

		// Breed the population with tournament selection and SCX crossover
		// Perform computation parallelized, build children iteratively
		for (unsigned int j = 1; j < node_quantity; ++j)
		{
			crossover << <grid, threads >> > (device_population, device_parents, device_state, device_cost_table, j);
			
			mutate << <grid, threads >> > (device_population, device_state, node_quantity);
			
			evaluatePopulation << <grid, threads >> > (device_population, device_cost_table, node_quantity);
		}
	}

	HANDLE_ERROR(hipEventRecord(stop));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipEventElapsedTime(&milliseconds, start, stop));

	// Copy memory back to host
	// TODO: Revisar si es necesaria
	HANDLE_ERROR(hipMemcpy(&initial_population, device_population, sizeof(population), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipDeviceSynchronize());
#pragma endregion

#pragma region OUTPUT

	/****************************************************************************************************
	* OUTPUT
	****************************************************************************************************/
	tour fittest = getFittestTour(initial_population.tours, POPULATION_SIZE);	
	printf("%f %f\n", milliseconds / 1000, fittest.total_distance);

#pragma endregion


	/****************************************************************************************************
	* FREE MEMORY
	****************************************************************************************************/
	HANDLE_ERROR(hipFree(d_node_matrix));
	HANDLE_ERROR(hipFree(d_node_t_matrix));
	free(h_node_t_matrix);
	HANDLE_ERROR(hipFree(d_distance));
	free(h_distance);
	free(matrix);
	free(i);
	free(n);
	free(d);

	HANDLE_ERROR(hipDeviceReset());
	// End Execution
	return 0;	
}