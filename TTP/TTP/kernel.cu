#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <vector>

#include <sstream>

#define DIMENSION "DIMENSION:"
#define ITEM_QTY "NUMBER OF ITEMS:"
#define KNAPSACK_CAPACITY "CAPACITY OF KNAPSACK:"
#define MIN_SPEED "MIN SPEED:"
#define MAX_SPEED "MAX SPEED:"
#define RENTING_RATIO "RENTING RATIO:"
#define EDGE_WEIGHT_TYPE "EDGE_WEIGHT_TYPE:"
#define NODE_COORD_SECTION "NODE_COORD_SECTION	(INDEX, X, Y):"
#define ITEMS_SECTION "ITEMS SECTION	(INDEX, PROFIT, WEIGHT, ASSIGNED NODE NUMBER):"

#define BLOCK_SIZE 16

const int blockPerGrid = 8;

void cudaCheckError()
{
	hipError_t e = hipGetLastError();
	if (e != hipSuccess)
	{
		fprintf(stderr, "CUDA Failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
	}
}

/// <summary>
/// Function to find a character position in a string
/// </summary>
/// <param name="stringToSearch">- String to search</param>
/// <param name="characterToFind">- Character to find in the string</param>
/// <returns>Position in the string of the character</returns>
size_t findCharacterPosition(char stringToSearch[], char characterToFind)
{
	size_t stringLength = 0, i, characterPosition = 0;

	stringLength = strlen(stringToSearch);
	for (i = 0; i < stringLength; i++)
	{
		if (stringToSearch[i] == characterToFind)
			characterPosition = i + 1;
	}

	return (characterPosition);
}

/// <summary>
/// Extracts a string from another string
/// </summary>
/// <param name="originalString">- Original string</param>
/// <param name="subString">- Resulting Substring</param>
/// <param name="position">- Initial position where the substring is about to begin</param>
/// <param name="length">- Length of the desired substring</param>
void subString(char originalString[], char subString[], size_t position, size_t length)
{
	int c = 0, d = 0;
	char tempSubString[1000];

	while (c < length) {
		subString[c] = originalString[position + c - 1];
		c++;
	}

	subString[c] = '\0';
	c = 0;

	while (subString[c] != '\0')
	{
		if (subString[c] == ' ' || subString[c] == '	') {
			int temp = c + 1;
			if (subString[temp] != '\0') {
				while ((subString[temp] == ' ' || subString[c] == '	') && subString[temp] != '\0') {
					if (subString[temp] == ' ' || subString[c] == '	')
						c++;
					temp++;
				}
			}
		}
		tempSubString[d] = subString[c];
		c++;
		d++;
	}

	tempSubString[d] = '\0';
	strcpy(subString, tempSubString);
}

/// <summary>
/// Function to count the amount of lines in a file
/// </summary>
/// <param name="fileName">- File path and name of the file to evaluate</param>
/// <returns>Amount of lines in the file</returns>
int countFileLines(char fileName[]) {

	FILE* filePtr;
	int lineCount = 0;
	char chr;

	filePtr = fopen(fileName, "r");
	chr = getc(filePtr);
	while (chr != EOF)
	{
		if (chr == '\n')
			lineCount++;
		chr = getc(filePtr);
	}
	fclose(filePtr);
	return lineCount;
}

/// <summary>
/// Count the rows for a matrix in a file with a given structure
/// </summary>
/// <param name="fileName">- File path and name of the file to evaluate</param>
/// <param name="sectionName">- Section name in the file where the matrix begins</param>
/// <returns>Amount of rows in the matrix</returns>
int countMatrixRows(const char fileName[], const char sectionName[])
{
	FILE* filePtr;
	char str[255], sub[255];
	int lineCount = 0, initialPosition = 0, rows = 0;
	const char openMode[] = "r";

	filePtr = fopen(fileName, openMode);

	while (fgets(str, 100, filePtr) != NULL) {
		if (strncmp(str, sectionName, strlen(sectionName)) == 0) {
			initialPosition = lineCount;
		}
		subString(str, sub, 1, 1);
		if (initialPosition != NULL && lineCount > initialPosition && isdigit(sub[0])) {
			rows++;
		}
		else if (initialPosition != NULL && lineCount > initialPosition && isalpha(sub[0]))
		{
			break;
		}
		lineCount++;
	}
	fclose(filePtr);
	return rows;
}

/// <summary>
/// Validates if a file exits
/// </summary>
/// <param name="path">- File path and name of the file</param>
/// <returns>0: File does not exist, 1: File exist</returns>
int fileExists(const char* path)
{
	// Try to open file
	FILE* fptr = fopen(path, "r");

	// If file doesn't exists
	if (fptr == NULL)
		return 0;

	// File exists hence close file and return true
	fclose(fptr);

	return 1;
}

/// <summary>
/// Extracts matrix from a file with a given structure
/// </summary>
/// <param name="fileName">- File path and name</param>
/// <param name="sectionName">- Section name in the file</param>
/// <param name="col">- Amount of columns</param>
/// <returns>- Double pointer matrix of integers</returns>
int** extractMatrix(const char fileName[], const char sectionName[], int col)
{
	FILE* filePtr;
	char str[255], sub[255], * token;
	int lineCount = 0, initialPosition = 0, rows, matrixRow, matrixCol;
	const char openMode[] = "r";

	filePtr = fopen(fileName, openMode);
	rows = countMatrixRows(fileName, sectionName);

	int** matrixResult = (int**)malloc(rows * sizeof(int));
	for (int i = 0; i < col; i++) {
		matrixResult[i] = (int*)malloc(sizeof(int) * col);
	}

	while (fgets(str, 100, filePtr) != NULL) {
		if (strncmp(str, sectionName, strlen(sectionName)) == 0) {
			initialPosition = lineCount;
		}
		subString(str, sub, 1, 1);
		if (initialPosition != NULL && lineCount > initialPosition && isdigit(sub[0])) {
			token = strtok(str, "	");
			matrixCol = 0;
			matrixRow = atoi(token) - 1;
			while (token != NULL)
			{
				matrixResult[matrixRow][matrixCol] = atoi(token);
				token = strtok(NULL, "	");
				if (matrixCol < col)
					matrixCol++;
			}
		}
		else if (initialPosition != NULL && lineCount > initialPosition && isalpha(sub[0]))
		{
			break;
		}
		lineCount++;
	}

	fclose(filePtr);

	return matrixResult;
}

/// <summary>
/// Extracts matrix from a file with a given structure
/// </summary>
/// <param name="fileName">- File path and name</param>
/// <param name="sectionName">- Section name in the file</param>
/// <param name="rows">- Amount of columns</param>
/// <param name="cols">- Amount of rows</param>
/// <returns>- Double pointer matrix of floats</returns>
float** extractMatrix(const char fileName[], const char sectionName[], int rows, int cols)
{
	FILE* filePtr;
	char str[255], sub[255], * token;
	int lineCount = 0, initialPosition = 0, matrixRow, matrixCol;
	const char openMode[] = "r";

	filePtr = fopen(fileName, openMode);

	// Allocate memory for rows
	float **matrixResult = (float**)malloc(rows * sizeof(float*));
	if (matrixResult == NULL) {
		fprintf(stderr, "Out of Memory");
		exit(0);
	}

	// Allocate memory for columns
	for (int i = 0; i < rows; i++) {
		matrixResult[i] = (float*)malloc(cols * sizeof(float));
		if (matrixResult[i] == NULL) {
			fprintf(stderr, "Out of Memory");
			exit(0);
		}
	}
	
	while (fgets(str, 100, filePtr) != NULL) {
		if (strncmp(str, sectionName, strlen(sectionName)) == 0) {
			initialPosition = lineCount;
		}
		subString(str, sub, 1, 1);
		if (initialPosition != NULL && lineCount > initialPosition && isdigit(sub[0])) {
			token = strtok(str, "	");
			matrixCol = 0;
			matrixRow = atoi(token) - 1;
			while (token != NULL)
			{
				matrixResult[matrixRow][matrixCol] = atoi(token);
				token = strtok(NULL, "	");
				if (matrixCol < cols)
					matrixCol++;
			}
		}
		else if (initialPosition != NULL && lineCount > initialPosition && isalpha(sub[0]))
		{
			break;
		}
		lineCount++;
	}

	fclose(filePtr);

	return matrixResult;
}

/// <summary>
/// Extracts matrix from a file with a given structure
/// </summary>
/// <param name="fileName">- File path and name</param>
/// <param name="sectionName">- Section name in the file</param>
/// <param name="rows">- Amount of columns</param>
/// <param name="cols">- Amount of rows</param>
/// <returns>Matrix pointer</returns>
float* extractMatrixFromFile(const char fileName[], const char sectionName[], int rows, int cols)
{
	FILE* file_ptr;
	char str[255], sub[255], * token;
	int lineCount = 0;
	int initialPosition = 0;
	int matrix_counter = 0;
	int matrix_size = rows * cols;
	const char openMode[] = "r";

	file_ptr = fopen(fileName, openMode);

	// Allocate memory for the result
	float* matrix_result = (float*)malloc(matrix_size * sizeof(float));
	if (matrix_result == NULL) {
		fprintf(stderr, "Out of Memory");
		exit(0);
	}

	while (fgets(str, 100, file_ptr) != NULL) {
		if (strncmp(str, sectionName, strlen(sectionName)) == 0) {
			initialPosition = lineCount;
		}
		subString(str, sub, 1, 1);
		if (initialPosition != NULL && lineCount > initialPosition && isdigit(sub[0])) {
			token = strtok(str, "	");
			while (token != NULL)
			{
				matrix_result[matrix_counter] = atoi(token);
				token = strtok(NULL, "	");
				if (matrix_counter < matrix_size)
					matrix_counter++;
			}
		}
		else if (initialPosition != NULL && lineCount > initialPosition && isalpha(sub[0]))
		{
			break;
		}
		lineCount++;
	}

	fclose(file_ptr);

	return matrix_result;
}

/// <summary>
/// Displays a matrix on screen
/// </summary>
/// <param name="matrix">- Matrix to display</param>
/// <param name="rows">- Amount of rows in the matrix</param>
/// <param name="columns">- Amount of columns in the matrix</param>
void display(int** matrix, int rows, int columns) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < columns; j++) {
			printf("%d ", matrix[i][j]);			
		}
		printf("\n");
	}
	printf("\n");
}

/// <summary>
/// Displays a matrix on screen
/// </summary>
/// <param name="matrix">- Matrix to display</param>
/// <param name="rows">- Amount of rows in the matrix</param>
/// <param name="columns">- Amount of columns in the matrix</param>
void display(float** matrix, int rows, int columns) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < columns; j++) {
			printf("%f ", matrix[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

/// <summary>
/// Displays a matrix on screen
/// </summary>
/// <param name="matrix">- Matrix to display</param>
/// <param name="rows">- Amount of rows in the matrix</param>
/// <param name="columns">- Amount of columns in the matrix</param>
void display(float* matrix, int rows, int columns) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < columns; j++) {
			printf("%f	", matrix[i * columns + j]);
		}
		printf("\n");
	}
	printf("\n");
}

/// <summary>
/// Calculates euclidean distance between a matrix of source points and a matrix of destination points
/// </summary>
/// <param name="srcPoint">- Matrix of source points</param>
/// <param name="dstPoint">- Matrix of destination points</param>
/// <param name="out">- Result matrix with distances</param>
/// <param name="rCount">- Row count</param>
/// <param name="size">- Total size of the result matrix</param>
void euclideanDistanceCPU(float** srcPoint, float** dstPoint, float** out, int rCount, int size) {
	for (int s = 0; s < size; s++) {
		for (int xSrc = 0; xSrc < rCount; xSrc++) {
			for (int xDst = 0; xDst < rCount; xDst++) {
				out[s][0] = (float)srcPoint[xSrc][0];
				out[s][1] = (float)dstPoint[xDst][0];
				out[s][2] = (float)sqrt(pow(dstPoint[xDst][1] - srcPoint[xSrc][1], 2) + pow(dstPoint[xDst][2] - srcPoint[xSrc][2], 2) * 1.0);
				s++;
			}
		}
	}
}

/// <summary>
/// Calculates euclidean distance between a matrix of source points and a matrix of destination points
/// </summary>
/// <param name="srcPoint">- Matrix of source points</param>
/// <param name="dstPoint">- Matrix of destination points</param>
/// <param name="out">- Result matrix with distances</param>
/// <param name="srcSize">- Size of the source matrix</param>
/// <param name="dstSize">- Size of the destination matrix</param>
/// <param name="cols">- Amount of columns</param>
void euclideanDistanceCPU(float* srcPoint, float* dstPoint, float* out, int srcSize, int dstSize, int cols) {
	int s = 0;
	for (int xSrc = 0; xSrc < srcSize; xSrc = xSrc + cols) {
		for (int xDst = 0; xDst < dstSize; xDst = xDst + cols) {
			out[s] = srcPoint[xSrc];
			out[s + 1] = dstPoint[xDst];
			out[s + 2] = sqrt(pow(dstPoint[xDst + 1] - srcPoint[xSrc + 1], 2) + pow(dstPoint[xDst + 2] - srcPoint[xSrc + 2], 2) * 1.0);
			s = s + cols;
		}
	}
}

/// <summary>
/// Function to remove the first column from a matrix
/// </summary>
/// <param name="matrix">Input Matrix</param>
void removeFirstColumn(float* matrix, float* out_matrix, int height, int width) {
	int width_out = width - 1;
	int pos = 0;
	int posM = 0;
	for (int i = 0; i < height; i++)
	{		
		for (int j = 0; j < width_out; j++)
		{
			pos = i * width_out + j;
			posM = i * width + j + 1;
			out_matrix[pos] = matrix[posM];
		}
	}
}

//TODO: Implement a function to remove any column from the matrix

/// <summary>
/// Kernel for matrix multiplication
/// </summary>
/// <param name="m_a_dev">- Input Matrix A</param>
/// <param name="m_b_dev">- Input Matrix B</param>
/// <param name="m_out_dev">- Output Matrix (Result of A X B)</param>
/// <param name="width">- Number of calculations per thread</param>
/// <param name="m_out_dev_rows">- Total of rows of the output matrix (Rows of Matrix A)</param>
/// <param name="m_out_dev_cols">- Total of columns of the output matrix (Columns of Matrix B)</param>
/// <returns></returns>
__global__ void matrixMultiplicationGPU(float* m_a_dev, float* m_b_dev, float* m_out_dev, int width, int m_out_dev_rows, int m_out_dev_cols) {
	
	/* Calculate global indexes*/
	unsigned int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

	// Check boundry conditions
	if (rowIdx < m_out_dev_rows && colIdx < m_out_dev_cols)
	{
		// Execute the multiplication for one row and one column
		float value = 0;
		for (int k = 0; k < width; k++)
		{
			value += m_a_dev[rowIdx * m_out_dev_rows + k] * m_b_dev[k * m_out_dev_cols + colIdx];
		}
		m_out_dev[rowIdx * m_out_dev_cols + colIdx] = value;
	}
}

/// <summary>
/// Kernel to calculate distances between point matrixes
/// </summary>
/// <param name="m_src_dev">- Matrix with source coodinates</param>
/// <param name="m_dst_dev">- Matrix with destination coordinates</param>
/// <param name="m_dist_dev">- Result Matrix with euclidean distances</param>
/// <param name="m_dist_dev_rows">- Result matrix row count</param>
/// <param name="m_dist_dev_cols">- Result matrix column count</param>
/// <returns></returns>
__global__ void matrixDistances(float* m_src_dev, float* m_dst_dev, float* m_dist_dev, int m_dist_dev_rows, int m_dist_dev_cols) {

	// Define variables
	const unsigned int width = 2;

	// Calculate global indexes
	unsigned int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

	// Check boundary conditions
	if (rowIdx < m_dist_dev_rows && colIdx < m_dist_dev_cols)
	{
		// Execute distance calculation
		float value = 0;
		for (int k = 0; k < width; k++)
		{
			value += pow(m_dst_dev[k * m_dist_dev_cols + colIdx] - m_src_dev[rowIdx * width + k], 2);
		}
		m_dist_dev[rowIdx * m_dist_dev_cols + colIdx] = sqrt(value);
	}
}

/// <summary>
/// Basic implementation of matrix transpose
/// </summary>
/// <param name="m_dev">- Matrix to be transposed on device memory</param>
/// <param name="t_m_dev">- Matrix Transpose result on device memory</param>
/// <param name="width">- Width of the matrix</param>
/// <param name="height">- Height of the matrix</param>
/// <returns></returns>
__global__ void matrixTranspose(float* m_dev, float* t_m_dev, int width, int height) {

	/* Calculate global index for this thread */
	unsigned int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

	/* Copy m_dev[rowIdx][colIdx] to t_m_dev[rowIdx][colIdx] */
	if (colIdx < width && rowIdx < height)
	{
		unsigned int index_in = colIdx + width * rowIdx;
		unsigned int index_out = rowIdx + height * colIdx;
		t_m_dev[index_out] = m_dev[index_in];
	}
}

/// <summary>
/// Optimized Kernel to ensure all global reads and writes are coalesced and to avoid bank conflicts in
/// shared memory. This Kernel is up to 11x faster than "matrix_transpose" kernel.
/// </summary>
/// <param name="m_dev">- Matrix to be transposed on device memory</param>
/// <param name="t_m_dev">- Matrix Transpose result on device memory</param>
/// <param name="width">- Width of the matrix</param>
/// <param name="height">- Height of the matrix</param>
/// <returns></returns>
__global__ void matrixTransposeCoalesced(float* m_dev, float* t_m_dev, int width, int height) {

	__shared__ float block[BLOCK_SIZE][BLOCK_SIZE + 1];

	// Read matrix tile into shared memory
	// Load one element per thread from device memory (m_dev) and store it in transposed order in block[][]
	unsigned int colIdx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	unsigned int rowIdx = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	if ((colIdx < width) && (rowIdx < height))
	{
		unsigned int index_in = rowIdx * width + colIdx;
		block[threadIdx.y][threadIdx.x] = m_dev[index_in];
	}

	// Synchronise to ensure allwrites to block[][] have completed
	__syncthreads();

	// Write the transposed matrix tile to global memory (t_m_dev) in linear order
	colIdx = blockIdx.y * BLOCK_SIZE + threadIdx.x;
	rowIdx = blockIdx.x * BLOCK_SIZE + threadIdx.y;
	if ((colIdx < height) && (rowIdx < width))
	{
		unsigned int index_out = rowIdx * height + colIdx;
		t_m_dev[index_out] = block[threadIdx.x][threadIdx.y];
	}
}

int main()
{
	char file_name[255], str[255], sub[255];
	FILE* fp;
	size_t position;
	const char openMode[] = "r";
	double Dimension, ItemQuantity, KnapsackCapacity, MinSpeed, MaxSpeed, RentingRatio;
	char EdgeWeightType[1000];

	// Ask for the filepath & name where the problem is defined
	printf("Enter name of a file you wish to see\n");
	gets_s(file_name);

	// Open the file in read mode
	fp = fopen(file_name, openMode);

	// Valida que no se presente algun error en la apertura del archivo
	if (fp == NULL)
	{
		perror("Error while opening the file. \n");
		exit(EXIT_FAILURE);
	}

	// Print headers
	printf("The contents of %s file are: \n", file_name);

	printf("The line quantity in file are: %d \n", countFileLines(file_name));

	// Obtain general data from file
	while (fgets(str, 100, fp) != NULL) {
		position = findCharacterPosition(str, ':');
		if (strncmp(str, DIMENSION, strlen(DIMENSION)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			Dimension = atof(sub);
			printf("Dimension is %lf \n", Dimension);
		}
		else if (strncmp(str, ITEM_QTY, strlen(ITEM_QTY)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			ItemQuantity = atof(sub);
			printf("Item Quantity is %lf \n", ItemQuantity);
		}
		else if (strncmp(str, KNAPSACK_CAPACITY, strlen(KNAPSACK_CAPACITY)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			KnapsackCapacity = atof(sub);
			printf("Knapsack Capacity is %lf \n", KnapsackCapacity);
		}
		else if (strncmp(str, MIN_SPEED, strlen(MIN_SPEED)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			MinSpeed = atof(sub);
			printf("Min Speed is %lf \n", MinSpeed);
		}
		else if (strncmp(str, MAX_SPEED, strlen(MAX_SPEED)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			MaxSpeed = atof(sub);
			printf("Max Speed is %lf \n", MaxSpeed);
		}
		else if (strncmp(str, RENTING_RATIO, strlen(RENTING_RATIO)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			RentingRatio = atof(sub);
			printf("Renting Ratio is %lf \n", RentingRatio);
		}
		else if (strncmp(str, EDGE_WEIGHT_TYPE, strlen(EDGE_WEIGHT_TYPE)) == 0)
		{
			subString(str, sub, position + 1, strlen(str) - position);
			strcpy(EdgeWeightType, sub);
			printf("Edge Weight Type is %s \n", EdgeWeightType);
		}
	}

	// Close file
	fclose(fp);
	
	// Obtain node matrix
	float* node_matrix;
	// Calculate amount of rows
	int node_rows = countMatrixRows(file_name, NODE_COORD_SECTION);
	// Calculate amount of columns
	int node_columns = 3;
	// Calculate node matrix size
	int node_matrix_size = node_columns * node_rows;
	// Get matrix
	node_matrix = extractMatrixFromFile(file_name, NODE_COORD_SECTION, node_rows, node_columns);
	// Visualize values for node matrix
	printf("Matrix of Nodes has %d rows \n\n", node_rows);
	printf("INDEX	X	Y\n");
	display(node_matrix, node_rows, node_columns);

	// Obtain item matrix
	float* item_matrix;
	// Calculate amount of rows
	int item_rows = countMatrixRows(file_name, ITEMS_SECTION);
	// Calculate amount of coluns
	int item_columns = 4;
	// Get matrix
	item_matrix = extractMatrixFromFile(file_name, ITEMS_SECTION, item_rows, item_columns);
	// Visualize values for item matrix
	printf("Matrix of items has %d rows \n\n", item_rows);
	printf("INDEX	PROFIT	WEIGHT	ASSIGNED NODE\n");
	display(item_matrix, item_rows, item_columns);

	// Calculate Distance Matrix in CPU
	float* distance_matrix;
	int distance_matrix_size = node_rows * node_rows * node_columns;
	distance_matrix = (float*)malloc(distance_matrix_size * sizeof(float));
	if (distance_matrix == NULL) {
		fprintf(stderr, "Out of Memory");
		exit(0);
	}

	euclideanDistanceCPU(node_matrix, node_matrix, distance_matrix, node_matrix_size, node_matrix_size, node_columns);
	printf("SOURCE	DESTINY	DISTANCE\n");
	display(distance_matrix, node_rows * node_rows, 3);	

	// Remove column
	// Define new matrix
	float* coordinate_matrix;
	int coordinate_matrix_size = node_rows * (node_columns - 1);
	coordinate_matrix = (float*)malloc(coordinate_matrix_size * sizeof(float));
	if (coordinate_matrix == NULL) {
		fprintf(stderr, "Out of Memory");
		exit(0);
	}
	removeFirstColumn(node_matrix, coordinate_matrix, node_rows, node_columns);
	printf("X	Y\n");
	display(coordinate_matrix, node_rows, node_columns - 1);

	// Calculate Distance Matrix in CUDA
	// Define device pointers
	float* d_distance_matrix;
	int distance_matrix_size_gpu = node_rows * node_rows * (node_columns-1);
	float* d_coordinate_matrix;
	float* d_coordinate_t_matrix;
	
	hipMalloc(&d_coordinate_matrix, sizeof(float) * coordinate_matrix_size);
	hipMalloc(&d_coordinate_t_matrix, sizeof(float) * coordinate_matrix_size);
	hipMemcpy(d_coordinate_matrix, coordinate_matrix, sizeof(float) * coordinate_matrix_size, hipMemcpyHostToDevice);

	// Setup execution parameters
	//dim3 grid(node_columns / BLOCK_SIZE, node_rows / BLOCK_SIZE, 1);
	dim3 grid(blockPerGrid, blockPerGrid, 1);
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 1);

	printf("Transponiendo la matrix de coordenadas de tama�o [%d][%d]\n", node_rows, node_columns-1);
	matrixTranspose << <grid, threads >> > (d_coordinate_matrix, d_coordinate_t_matrix, node_columns-1, node_rows);
	hipDeviceSynchronize();

	//Copy results from device to host
	float* h_coordinate_t_matrix = (float*)malloc(sizeof(float) * coordinate_matrix_size);
	hipMemcpy(h_coordinate_t_matrix, d_coordinate_t_matrix, sizeof(float) * coordinate_matrix_size, hipMemcpyDeviceToHost);

	display(h_coordinate_t_matrix, node_columns-1, node_rows);

	hipMalloc(&d_distance_matrix, sizeof(float)* distance_matrix_size_gpu);
	printf("Calculando la matriz de distancias\n");
	matrixDistances << <grid, threads >> > (d_coordinate_matrix, d_coordinate_t_matrix, d_distance_matrix, node_rows, node_rows);
	hipDeviceSynchronize();

	//Copy results from device to host
	float* h_distance_matrix = (float*)malloc(sizeof(float) * distance_matrix_size_gpu);
	hipMemcpy(h_distance_matrix, d_distance_matrix, sizeof(float)* distance_matrix_size_gpu, hipMemcpyDeviceToHost);

	display(h_distance_matrix, node_rows, node_rows);

	hipFree(d_distance_matrix);
	hipFree(d_coordinate_matrix);
	hipFree(d_coordinate_t_matrix);
	free(coordinate_matrix);
	free(h_coordinate_t_matrix);
	free(distance_matrix);
	free(node_matrix);
	free(item_matrix);
	
	/*Initialize CUDA Sub Routines*/
	int count;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&count);
	printf("**********************************************************************************************\n");
	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("GPU: %s\n", prop.name);
		printf("Compute Mode: %d\n", prop.computeMode);
		printf("Max Grid Size: %d\n", prop.maxGridSize);
		printf("Warp Size: %d\n", prop.warpSize);
		printf("Total Global Memory: %zd\n", prop.totalGlobalMem);
		printf("Total Constant Memory: %zd\n", prop.totalConstMem);
		printf("Shared Memory Per Block: %zd\n", prop.sharedMemPerBlock);
		printf("Multiprocessor: %d\n", prop.multiProcessorCount);
		printf("Max Threads Per Multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
		printf("Max Blocks Per Multiprocessor: %d\n", prop.maxBlocksPerMultiProcessor);
		printf("Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
		printf("Max Size of Each Dimension of a Block: %d\n", prop.maxThreadsDim);
	}
	printf("**********************************************************************************************\n");

	// End Execution
	return 0;
}