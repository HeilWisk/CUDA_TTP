#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <vector>
#include <time.h>
#include <chrono>

#include "headers/helper_functions.h"
#include "headers/hip/hip_runtime_api.h"
#include "headers/config.h"
#include "headers/item.cuh"
#include "headers/node.cuh"
#include "headers/params.h"
#include "headers/distance.cuh"
#include "headers/greedy.h"
#include "headers/tour.cuh"
#include "headers/population.cuh"
#include "headers/genetic.cuh"
#include "headers/File.h"

#define NAME "PROBLEM NAME:"
#define DIMENSION "DIMENSION:"
#define ITEM_QTY "NUMBER OF ITEMS:"
#define KNAPSACK_CAPACITY "CAPACITY OF KNAPSACK:"
#define MIN_SPEED "MIN SPEED:"
#define MAX_SPEED "MAX SPEED:"
#define RENTING_RATIO "RENTING RATIO:"
#define EDGE_WEIGHT_TYPE "EDGE_WEIGHT_TYPE:"
#define NODE_COORD_SECTION "NODE_COORD_SECTION	(INDEX, X, Y):"
#define ITEMS_SECTION "ITEMS SECTION	(INDEX, PROFIT, WEIGHT, ASSIGNED NODE NUMBER):"

#pragma region GLOBAL VARIABLES

/// Declare structs Globally
/// when the program is configured with big instances (I.E: Cities > 50) the stack pile generates error
/// that's because when an array is declared locally , then it always initializes in the stack memory
/// and generally, this memory has a size limit. When an array is declared globally then it stores in
/// the data segment and it has no size limit.



#pragma endregion

/// <summary>
/// Convert a char to uppercase
/// </summary>
/// <param name="fileName"></param>
/// <param name="name"></param>
void toUpperCase(char fileName[])
{
	int q = 0;
	char ch;

	while (fileName[q])
	{
		ch = fileName[q];
		fileName[q] = toupper(ch);
		q++;
	}
}

/// <summary>
/// Function to count the amount of lines in a file
/// </summary>
/// <param name="fileName">- File path and name of the file to evaluate</param>
/// <returns>Amount of lines in the file</returns>
int countFileLines(char fileName[]) {

	FILE* filePtr;
	int lineCount = 0;
	char chr;

	filePtr = fopen(fileName, "r");
	chr = getc(filePtr);
	while (chr != EOF)
	{
		if (chr == '\n')
			lineCount++;
		chr = getc(filePtr);
	}
	fclose(filePtr);
	return lineCount;
}

/// <summary>
/// Function to find a character position in a string
/// </summary>
/// <param name="stringToSearch">- String to search</param>
/// <param name="characterToFind">- Character to find in the string</param>
/// <returns>Position in the string of the character</returns>
size_t findCharacterPosition(char stringToSearch[], char characterToFind)
{
	size_t stringLength = 0, i, characterPosition = 0;

	stringLength = strlen(stringToSearch);
	for (i = 0; i < stringLength; i++)
	{
		if (stringToSearch[i] == characterToFind)
			characterPosition = i + 1;
	}

	return (characterPosition);
}

/// <summary>
/// Extracts a string from another string
/// </summary>
/// <param name="originalString">- Original string</param>
/// <param name="subString">- Resulting Substring</param>
/// <param name="position">- Initial position where the substring is about to begin</param>
/// <param name="length">- Length of the desired substring</param>
void subString(char originalString[], char subString[], size_t position, size_t length)
{
	int c = 0, d = 0;
	char tempSubString[1000];

	while (c < length) {
		subString[c] = originalString[position + c - 1];
		c++;
	}

	subString[c] = '\0';
	c = 0;

	while (subString[c] != '\0')
	{
		if (subString[c] == ' ' || subString[c] == '	') {
			int temp = c + 1;
			if (subString[temp] != '\0') {
				while ((subString[temp] == ' ' || subString[c] == '	') && subString[temp] != '\0') {
					if (subString[temp] == ' ' || subString[c] == '	')
						c++;
					temp++;
				}
			}
		}
		tempSubString[d] = subString[c];
		c++;
		d++;
	}

	tempSubString[d] = '\0';
	strcpy(subString, tempSubString);
}

/// <summary>
/// Count the rows for a matrix in a file with a given structure
/// </summary>
/// <param name="fileName">- File path and name of the file to evaluate</param>
/// <param name="sectionName">- Section name in the file where the matrix begins</param>
/// <returns>Amount of rows in the matrix</returns>
int countMatrixRows(const char fileName[], const char sectionName[])
{
	FILE* filePtr;
	char str[255], sub[255];
	int lineCount = 0, initialPosition = 0, rows = 0;
	const char openMode[] = "r";

	filePtr = fopen(fileName, openMode);

	while (fgets(str, 100, filePtr) != NULL) {
		if (strncmp(str, sectionName, strlen(sectionName)) == 0) {
			initialPosition = lineCount;
		}
		subString(str, sub, 1, 1);
		if (initialPosition != NULL && lineCount > initialPosition && isdigit(sub[0])) {
			rows++;
		}
		else if (initialPosition != NULL && lineCount > initialPosition && isalpha(sub[0]))
		{
			break;
		}
		lineCount++;
	}
	fclose(filePtr);
	return rows;
}

/// <summary>
/// Extracts matrix from a file with a given structure
/// </summary>
/// <param name="fileName">- File path and name</param>
/// <param name="sectionName">- Section name in the file</param>
/// <param name="rows">- Amount of columns</param>
/// <param name="cols">- Amount of rows</param>
/// <returns>Double pointer matrix of ints</returns>
int** extractMatrixFromFile(const char fileName[], const char sectionName[], int rows, int cols)
{
	FILE* filePtr;
	char str[255], sub[255], * token;
	int lineCount = 0, initialPosition = 0, matrixRow, matrixCol;
	const char openMode[] = "r";

	filePtr = fopen(fileName, openMode);

	// Allocate memory for rows
	int** matrixResult = (int**)malloc(rows * sizeof(int*));
	if (matrixResult == NULL) {
		fprintf(stderr, "Out of Memory");
		exit(0);
	}

	// Allocate memory for columns
	for (int i = 0; i < rows; i++) {
		matrixResult[i] = (int*)malloc(cols * sizeof(int));
		if (matrixResult[i] == NULL) {
			fprintf(stderr, "Out of Memory");
			exit(0);
		}
	}

	while (fgets(str, 100, filePtr) != NULL) {
		if (strncmp(str, sectionName, strlen(sectionName)) == 0) {
			initialPosition = lineCount;
		}
		subString(str, sub, 1, 1);
		if (initialPosition != NULL && lineCount > initialPosition && isdigit(sub[0])) {
			token = strtok(str, "	");
			matrixCol = 0;
			matrixRow = atoi(token) - 1;
			while (token != NULL)
			{
				matrixResult[matrixRow][matrixCol] = atoi(token);
				token = strtok(NULL, "	");
				if (matrixCol < cols)
					matrixCol++;
			}
		}
		else if (initialPosition != NULL && lineCount > initialPosition && isalpha(sub[0]))
		{
			break;
		}
		lineCount++;
	}

	fclose(filePtr);

	return matrixResult;
}

/// <summary>
/// Calculates euclidean distance between a matrix of source points and a matrix of destination points
/// </summary>
/// <param name="srcPoint">- Matrix of source points</param>
/// <param name="dstPoint">- Matrix of destination points</param>
/// <param name="out">- Result matrix with distances</param>
/// <param name="rCount">- Row count</param>
/// <param name="size">- Total size of the result matrix</param>
void euclideanDistanceCPU(node* srcPoint, node* dstPoint, distance* out, int rCount, int size) {
	for (int s = 0; s < size; s++) {
		for (int xSrc = 0; xSrc < rCount; xSrc++) {
			for (int xDst = 0; xDst < rCount; xDst++) {
				out[s].source = srcPoint[xSrc].id;
				out[s].destiny = dstPoint[xDst].id;
				out[s].value = (float)sqrt(pow(dstPoint[xDst].x - srcPoint[xSrc].x, 2) + pow(dstPoint[xDst].y - srcPoint[xSrc].y, 2) * 1.0);
				s++;
			}
		}
	}
}

/// <summary>
/// 
/// </summary>
/// <param name="state"></param>
/// <param name="seed"></param>
/// <returns></returns>
__global__ void initCuRand(hiprandState* state, time_t seed)
{
	// Calculate global index of the threads for the 2D GRID
	// Global index of every block on the grid
	int block_number_in_grid = blockIdx.x + gridDim.x * blockIdx.y;
	// Global index of every thread in block
	int thread_number_in_block = threadIdx.x + blockDim.x * threadIdx.y;
	// Number of thread per block
	int threads_per_block = blockDim.x * blockDim.y;
	// Global index of every thread on the grid
	int thread_global_index = block_number_in_grid * threads_per_block + thread_number_in_block;

	if (thread_global_index >= TOURS)
		return;

	hiprand_init(seed, thread_global_index, 0, &state[thread_global_index]);
}

/// <summary>
/// 
/// </summary>
/// <param name="tour"></param>
/// <param name="tour_size"></param>
/// <returns></returns>
__global__ void tourTest(tour* tour, int tour_size)
{
	for (int t = 0; t < tour_size; ++t)
	{
		printf(" > tour[%d].fitness: %f\n", t, tour[t].fitness);
		printf(" > tour[%d].total_distance: %f\n", t, tour[t].total_distance);
		for (int n = 0; n < CITIES; ++n)
		{
			printf(" > tour[%d].nodes[%d].id: %d\n", t, n, tour[t].nodes[n].id);
			printf(" > tour[%d].nodes[%d].x: %lf\n", t, n, tour[t].nodes[n].x);
			printf(" > tour[%d].nodes[%d].y: %lf\n", t, n, tour[t].nodes[n].y);
			for (int i = 0; i < ITEMS; ++i)
			{
				printf(" > tour[%d].nodes[%d].items[%d].id: %d\n", t, n, i, tour[t].nodes[n].items[i].id);
				printf(" > tour[%d].nodes[%d].items[%d].node: %d\n", t, n, i, tour[t].nodes[n].items[i].node);
				printf(" > tour[%d].nodes[%d].items[%d].value: %d\n", t, n, i, tour[t].nodes[n].items[i].value);
				printf(" > tour[%d].nodes[%d].items[%d].weight: %d\n", t, n, i, tour[t].nodes[n].items[i].weight);
			}
		}
	}
	printf("\n\n");
}

/// <summary>
/// 
/// </summary>
/// <param name="population"></param>
/// <returns></returns>
__global__ void populationTest(population* population)
{
	for (int p = 0; p < POPULATION_SIZE; ++p)
	{
		for (int t = 0; t < TOURS; ++t)
		{
			printf(" > population[%d].tours[%d].fitness: %f\n", p, t, population[p].tours[t].fitness);
			printf(" > population[%d].tours[%d].total_distance: %f\n", p, t, population[p].tours[t].total_distance);
			for (int n = 0; n < CITIES; ++n)
			{
				if (population[p].tours[t].nodes[n].id > 0)
				{
					printf(" > population[%d].tours[%d].nodes[%d].id: %d\n", p, t, n, population[p].tours[t].nodes[n].id);
					printf(" > population[%d].tours[%d].nodes[%d].x: %lf\n", p, t, n, population[p].tours[t].nodes[n].x);
					printf(" > population[%d].tours[%d].nodes[%d].y: %lf\n", p, t, n, population[p].tours[t].nodes[n].y);
					for (int i = 0; i < ITEMS; ++i)
					{
						if (population[p].tours[t].nodes[n].items[i].id >= 0)
						{
							printf(" > population[%d].tours[%d].nodes[%d].items[%d].id: %d\n", p, t, n, i, population[p].tours[t].nodes[n].items[i].id);
							printf(" > population[%d].tours[%d].nodes[%d].items[%d].node: %d\n", p, t, n, i, population[p].tours[t].nodes[n].items[i].node);
							printf(" > population[%d].tours[%d].nodes[%d].items[%d].value: %d\n", p, t, n, i, population[p].tours[t].nodes[n].items[i].value);
							printf(" > population[%d].tours[%d].nodes[%d].items[%d].weight: %d\n", p, t, n, i, population[p].tours[t].nodes[n].items[i].weight);
						}
					}
				}
			}
		}
	}
	printf("\n\n");
}

/// <summary>
/// 
/// </summary>
/// <param name="initial_population"></param>
/// <param name="distances"></param>
/// <param name="node_quantity"></param>
/// <param name="item_quantity"></param>
/// <param name="state"></param>
/// <returns></returns>
__global__ void initPopulationGPU(population* initial_population, tour* initial_tour, const int population_size, hiprandState* state)
{
	node temp;

	// Get thread ID
	// Global index of every block on the grid
	unsigned int block_number_in_grid = blockIdx.x + gridDim.x * blockIdx.y;
	// Global index of every thread in block
	unsigned int thread_number_in_block = threadIdx.x + blockDim.x * threadIdx.y;
	// Number of thread per block
	unsigned int threads_per_block = blockDim.x * blockDim.y;
	// Global index of every thread on the grid
	unsigned int thread_global_index = block_number_in_grid * threads_per_block + thread_number_in_block;

	hiprandState local_state = state[thread_global_index];

	// Set the tours
	for (int p = 0; p < population_size; ++p)
	{
		for (int n = 0; n < CITIES; ++n)
		{
			initial_population->tours[thread_global_index].nodes[n] = initial_tour[p].nodes[n];
		}
	}

	for (int j = 1; j < CITIES; ++j)
	{
		int random_position = 1 + (hiprand(&local_state) % (CITIES - 1));

		temp = initial_population->tours[thread_global_index].nodes[j];
		for (int k = 0; k < ITEMS; ++k)
		{
			temp.items[k] = initial_population->tours[thread_global_index].nodes[j].items[k];
		}

		printf(" > thread_global_index: %d > %d cambia con %d\n", thread_global_index, j, random_position);
		printf(" > thread_global_index: %d > El id de %d es: %d\n", thread_global_index, j, initial_population->tours[thread_global_index].nodes[j].id);
		printf(" > thread_global_index: %d > El id de %d es: %d\n", thread_global_index, random_position, initial_population->tours[thread_global_index].nodes[random_position].id);

		initial_population->tours[thread_global_index].nodes[j] = initial_population->tours[thread_global_index].nodes[random_position];
		for (int l = 0; l < ITEMS; ++l)
		{
			initial_population->tours[thread_global_index].nodes[j].items[l] = initial_population->tours[thread_global_index].nodes[random_position].items[l];
		}

		initial_population->tours[thread_global_index].nodes[random_position] = temp;
		for (int m = 0; m < ITEMS; ++m)
		{
			initial_population->tours[thread_global_index].nodes[random_position].items[m] = temp.items[m];
		}

		printf(" > thread_global_index: %d > initial_population->tours[%d].nodes[%d]: %d\n", thread_global_index, thread_global_index, j, initial_population->tours[thread_global_index].nodes[j].id);
		printf(" > thread_global_index: %d > initial_population->tours[%d].nodes[%d]: %d\n", thread_global_index, thread_global_index, random_position, initial_population->tours[thread_global_index].nodes[random_position].id);
	}
}

/// <summary>
/// Basic implementation of matrix transpose
/// </summary>
/// <param name="m_dev">- Matrix to be transposed on device memory</param>
/// <param name="t_m_dev">- Matrix Transpose result on device memory</param>
/// <param name="width">- Width of the matrix</param>
/// <param name="height">- Height of the matrix</param>
/// <returns></returns>
__global__ void transpose(node* m_dev, node* t_m_dev, int width, int height) {

	/* Calculate global index for this thread */
	unsigned int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

	/* Copy m_dev[rowIdx][colIdx] to t_m_dev[rowIdx][colIdx] */
	if (colIdx < width && rowIdx < height)
	{
		unsigned int index_in = colIdx + width * rowIdx;
		unsigned int index_out = rowIdx + height * colIdx;
		t_m_dev[index_out] = m_dev[index_in];
		for (int i = 0; i > ITEMS; ++i)
		{
			t_m_dev[index_out].items[i] = m_dev[index_in].items[i];
		}
	}
}

/// <summary>
/// Kernel to calculate distances between point matrixes
/// </summary>
/// <param name="m_src_dev">- Matrix with source coodinates</param>
/// <param name="m_dst_dev">- Matrix with destination coordinates</param>
/// <param name="m_dist_dev">- Result Matrix with euclidean distances</param>
/// <param name="m_dist_dev_rows">- Result matrix row count</param>
/// <param name="m_dist_dev_cols">- Result matrix column count</param>
/// <returns></returns>
__global__ void matrixDistances(node* m_src_dev, node* m_dst_dev, distance* m_dist_dev, int m_dist_dev_rows, int m_dist_dev_cols) {

	// Define variables
	const unsigned int width = 1;

	// Calculate global indexes
	unsigned int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

	// Check boundary conditions
	if (rowIdx < m_dist_dev_rows && colIdx < m_dist_dev_cols)
	{
		// Execute distance calculation
		float value = 0;
		int sourceId = 0;
		int destinyId = 0;
		for (int k = 0; k < width; k++)
		{
			sourceId = m_src_dev[rowIdx * width + k].id;
			destinyId = m_dst_dev[k * m_dist_dev_cols + colIdx].id;
			value += pow(m_dst_dev[k * m_dist_dev_cols + colIdx].x - m_src_dev[rowIdx * width + k].x, 2) + pow(m_dst_dev[k * m_dist_dev_cols + colIdx].y - m_src_dev[rowIdx * width + k].y, 2);
		}
		m_dist_dev[rowIdx * m_dist_dev_cols + colIdx].source = sourceId;
		m_dist_dev[rowIdx * m_dist_dev_cols + colIdx].destiny = destinyId;
		m_dist_dev[rowIdx * m_dist_dev_cols + colIdx].value = sqrt(value);
	}
}

/// <summary>
/// 
/// </summary>
/// <param name="population"></param>
/// <param name="distanceTable"></param>
/// <returns></returns>
__global__ void evaluatePopulation(population* population, distance* distanceTable)
{
	// Get Thread (particle) ID
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid >= TOURS)
		return;

	evaluateTour(population->tours[tid], distanceTable);
}

/// <summary>
/// 
/// </summary>
/// <param name="population"></param>
/// <param name="distanceTable"></param>
/// <returns></returns>
__global__ void evaluatePopulation(population* population, parameters* problem_parameters)
{
	// Calculate global index of the threads for the 2D GRID
	// Global index of every block on the grid
	int block_number_in_grid = blockIdx.x + gridDim.x * blockIdx.y;
	// Global index of every thread in block
	int thread_number_in_block = threadIdx.x + blockDim.x * threadIdx.y;
	// Number of thread per block
	int threads_per_block = blockDim.x * blockDim.y;
	// Global index of every thread on the grid
	int thread_global_index = block_number_in_grid * threads_per_block + thread_number_in_block;

	if (thread_global_index >= TOURS)
		return;

	evaluateTour(population->tours[thread_global_index], problem_parameters);
}


//void executeGeneticParallel(parameters params, char* name, int iterationCount, int executionCounter, double initialPopulationTimer, population* device_population, tour* device_parents, tour* device_offspring, parameters* device_parameters, hiprandState* device_states)
//{
	// Define CUDA Timers
//	hipEvent_t startKernel;
//	hipEvent_t stopKernel;	

	// Define variables for time (ms) counters
//	float elapsedSelectionGPU;
//	float elapsedCrossoverGPU;
//	float elapsedLocalSearchGPU;

	// Define number of threads and blocks
//	dim3 threadsPerBlock(THREADS_X, THREADS_Y);
//	dim3 numBlocks(THREADS / threadsPerBlock.x, THREADS / threadsPerBlock.y);

//	hipError_t hipError_t = hipSuccess;
	
//	checkCudaErrors(hipEventCreate(&startKernel));
//	checkCudaErrors(hipEventCreate(&stopKernel));

	// Select Parents For The Next Generation
//	checkCudaErrors(hipEventRecord(startKernel, 0));
//	selectionKernel << <numBlocks, threadsPerBlock >> > (device_population, device_parents, device_states);
//	hipError_t = hipGetLastError();
//	if (hipError_t != hipSuccess)
//	{
//		fprintf(stderr, "Selection Kernel: %s\n", hipGetErrorString(hipError_t));
//		exit(0);
//	}
//	checkCudaErrors(hipEventRecord(stopKernel, 0));
//	checkCudaErrors(hipEventSynchronize(stopKernel));
//	checkCudaErrors(hipEventElapsedTime(&elapsedSelectionGPU, startKernel, stopKernel));
	
	// Breed the population performing crossover (Combination of Ordered Crossover 
	// for the TSP sub-problem and One Point Crossover for the KP sub-problem)
//	checkCudaErrors(hipEventRecord(startKernel, 0));
//	crossoverKernel << <numBlocks, threadsPerBlock >> > (device_population, device_parents, device_offspring, device_parameters, device_states);
//	hipError_t = hipGetLastError();
//	if (hipError_t != hipSuccess)
//	{
//		fprintf(stderr, "Crossover Kernel: %s\n", hipGetErrorString(hipError_t));
//		exit(0);
//	}
//	checkCudaErrors(hipEventRecord(stopKernel, 0));
//	checkCudaErrors(hipEventSynchronize(stopKernel));
//	checkCudaErrors(hipEventElapsedTime(&elapsedCrossoverGPU, startKernel, stopKernel));
	
	// Perform local search (mutation)
//	checkCudaErrors(hipEventRecord(startKernel, 0));
//	localSearchKernel << <numBlocks, threadsPerBlock >> > (device_population, device_parameters, device_states);
//	hipError_t = hipGetLastError();
//	if (hipError_t != hipSuccess)
//	{
//		fprintf(stderr, "Local Search Kernel: %s\n", hipGetErrorString(hipError_t));
//		exit(0);
//	}
//	checkCudaErrors(hipEventRecord(stopKernel, 0));
//	checkCudaErrors(hipEventSynchronize(stopKernel));
//	checkCudaErrors(hipEventElapsedTime(&elapsedLocalSearchGPU, startKernel, stopKernel));
	
	// Copy Device Information to Host
//	checkCudaErrors(hipMemcpy(&initial_population_gpu, device_population, sizeof(population), hipMemcpyDeviceToHost));
//	checkCudaErrors(hipDeviceSynchronize());
	
	// Get Fittest tour of the generation
//	fittestOnEarth = getFittestTour(initial_population_gpu.tours, TOURS);
//	saveFittest(name, fittestOnEarth, params, iterationCount + 1, CUDA, executionCounter);
//	saveStatistics(name, CUDA, executionCounter, iterationCount + 1, initialPopulationTimer, elapsedSelectionGPU, elapsedCrossoverGPU, elapsedLocalSearchGPU);

//	checkCudaErrors(hipEventDestroy(startKernel));
//	checkCudaErrors(hipEventDestroy(stopKernel));
//}

/// <summary>
/// Method to execute the genetic algoritm in CPU
/// </summary>
/// <param name="problem"></param>
/// <param name="name"></param>
/// <param name="iterationCount"></param>
/// <param name="executionCounter"></param>
/// <param name="initialPopulationTimer"></param>
//void executeGeneticSequential(parameters problem, char* name, int iterationCount, int executionCounter, double initialPopulationTimer)
//{
	// Define timers
//	struct timespec startMethod;
//	struct timespec stopMethod;

	// Define variables for the time (ms) counters
//	double elapsedSelectionCPU;
//	double elapsedCrossoverCPU;
//	double elapsedLocalSearchCPU;

	// Select the best parents of the current generation
//	if (timespec_get(&startMethod, TIME_UTC) != TIME_UTC)
//	{
//		printf("Error in calling timespec_get\n");
//		exit(EXIT_FAILURE);
//	}
//	selection(initial_population_cpu, host_parents);
//	if (timespec_get(&stopMethod, TIME_UTC) != TIME_UTC)
//	{
//		printf("Error in calling timespec_get\n");
//		exit(EXIT_FAILURE);
//	}
//	elapsedSelectionCPU = (double)(stopMethod.tv_sec - startMethod.tv_sec) + ((double)(stopMethod.tv_nsec - startMethod.tv_nsec) * 1.e-6);

	// Breed the population performing crossover (Combination of Ordered Crossover 
	// for the TSP sub-problem and One Point Crossover for the KP sub-problem)
//	if (timespec_get(&startMethod, TIME_UTC) != TIME_UTC)
//	{
//		printf("Error in calling timespec_get\n");
//		exit(EXIT_FAILURE);
//	}
//	crossover(initial_population_cpu, host_parents, problem);
//	if (timespec_get(&stopMethod, TIME_UTC) != TIME_UTC)
//	{
//		printf("Error in calling timespec_get\n");
//		exit(EXIT_FAILURE);
//	}
//	elapsedCrossoverCPU = (double)(stopMethod.tv_sec - startMethod.tv_sec) + ((double)(stopMethod.tv_nsec - startMethod.tv_nsec) * 1.e-6);

//	if (timespec_get(&startMethod, TIME_UTC) != TIME_UTC)
//	{
//		printf("Error in calling timespec_get\n");
//		exit(EXIT_FAILURE);
//	}
//	localSearch(initial_population_cpu, problem);
//	if (timespec_get(&stopMethod, TIME_UTC) != TIME_UTC)
//	{
//		printf("Error in calling timespec_get\n");
//		exit(EXIT_FAILURE);
//	}
//	elapsedLocalSearchCPU = (double)(stopMethod.tv_sec - startMethod.tv_sec) + ((double)(stopMethod.tv_nsec - startMethod.tv_nsec) * 1.e-6);

	// Get Fittest tour of the generation
//	fittestOnEarth = getFittestTour(initial_population_cpu.tours, TOURS);
//	saveFittest(name, fittestOnEarth, problem, iterationCount + 1, NO_CUDA, executionCounter);
//	saveStatistics(name, NO_CUDA, executionCounter, iterationCount + 1, initialPopulationTimer, elapsedSelectionCPU, elapsedCrossoverCPU, elapsedLocalSearchCPU);
//}

int main()
{

#pragma region VARIABLES
	/****************************************************************************************************
	* DECLARE VARIABLES
	****************************************************************************************************/

	// File variables
	char file_name[255];
	char fname[_MAX_FNAME];
	char str[255];
	char sub[255];
	FILE* file_open;
	size_t position;
	const char openMode[] = "r";

	// Big File Variables
	char lines[9][255];
	unsigned int line_count = 0;
	char* linePointer;

	// Problem params
	parameters problem;
	int** matrix;
	char edge_weight_type[255];

	unsigned int population_size = POPULATION_SIZE;
	unsigned int tour_size = TOURS;

	tour initial_tour;
	population initial_population_cpu;
	population initial_population_gpu;
	tour fittestOnEarth;

	tour host_parents[SELECTED_PARENTS];
	tour host_tournament[TOURNAMENT_SIZE];

	int deviceCount = 0;
	hipDeviceProp_t properties;
	hipError_t deviceErr;
	hipError_t err = hipSuccess;

	// Device Variables
	population* device_population;
	tour* device_initial_tour;
	tour* device_parents;
	tour* device_offspring;
	parameters* device_parameters;
	hiprandState* device_states;
	//node* device_node_matrix;
	//node* device_node_t_matrix;
	//distance* device_distance;

	// CPU Timers
	struct timespec startMethod;
	struct timespec stopMethod;
	std::chrono::high_resolution_clock::time_point cpu_method_start;
	std::chrono::high_resolution_clock::time_point cpu_method_stop;

	struct timespec startCPU;
	struct timespec stopCPU;
	std::chrono::high_resolution_clock::time_point cpu_start;
	std::chrono::high_resolution_clock::time_point cpu_stop;

	double elapsedTimeInitialPopulationCPU;
	double elapsedTimeCPU;

	double elapsedSelectionCPU;
	double elapsedCrossoverCPU;
	double elapsedLocalSearchCPU;

	// GPU Timers
	float gpuExecutionTime = 0.0;
	hipEvent_t startKernel;
	hipEvent_t stopKernel;

	hipEvent_t startGPU;
	hipEvent_t stopGPU;
	
	float elapsedTimeInitialPopulationGPU;
	float elapsedTimeGPU;

	// Kernel Execution Parameters
	dim3 threadsPerBlock(THREADS_X, THREADS_Y);
	dim3 numBlocks(THREADS / threadsPerBlock.x, THREADS / threadsPerBlock.y);

	float elapsedSelectionGPU;
	float elapsedCrossoverGPU;
	float elapsedLocalSearchGPU;

#pragma endregion

#pragma region CAPTURE FILE PATH

	/****************************************************************************************************
	* CAPTURE FILE PATH AND LOAD HIS DATA
	****************************************************************************************************/
	// Ask for the filepath & name where the problem is defined
	printf("Enter name of a file you wish to see\n");
	gets_s(file_name);
	printf("\n");

	// Open the file in read mode
	file_open = fopen(file_name, openMode);

	// Validates for errors on file opening
	if (file_open == NULL)
	{
		perror("Error while opening the file.\n");
		exit(EXIT_FAILURE);
	}

	while (fgets(lines[line_count], 100, file_open) != NULL)
	{
		printf("line[%06d]: %s", line_count, lines[line_count]);
		if (lines[line_count][strlen(lines[line_count]) - 1] != '\n')
			printf("\n");
		else

		++line_count;
	}

	fclose(file_open);

	for (int i = 0; i < 9; i++)
	{
		linePointer = lines[i];
		linePointer[strlen(linePointer) - 1] = '\0';
		// Open the file in read mode
		file_open = fopen(lines[i], openMode);

		// Validates for errors on file opening
		if (file_open == NULL)
		{
			perror("Error while opening the file.\n");
			exit(EXIT_FAILURE);
		}

		_splitpath(lines[i], NULL, NULL, fname, NULL);
		toUpperCase(fname);

		// Print headers
		printf("****************************************************************************************\n");
		printf("CONTENTS OF THE FILE:\n");
		printf("****************************************************************************************\n");
		printf("The quantity of lines in the file are:	%d\n", countFileLines(lines[i]));

		// Obtain general data from file
		while (fgets(str, 100, file_open) != NULL) {
			position = findCharacterPosition(str, ':');
			// Extract problem name
			if (strncmp(str, NAME, strlen(NAME)) == 0)
			{
				subString(str, sub, position + 1, strlen(str) - position - 1);
				strcpy(problem.name, sub);
			}
			// Extract amount of nodes (cities)
			if (strncmp(str, DIMENSION, strlen(DIMENSION)) == 0)
			{
				subString(str, sub, position + 1, strlen(str) - position);
				problem.cities_amount = atoi(sub);
				printf("Nodes (Cities):				%d\n", problem.cities_amount);
			}
			// Extract the amount of items
			else if (strncmp(str, ITEM_QTY, strlen(ITEM_QTY)) == 0)
			{
				subString(str, sub, position + 1, strlen(str) - position);
				problem.items_amount = atoi(sub);
				problem.items_per_city = problem.items_amount / (problem.cities_amount - 1);
				printf("Item:					%d\n", problem.items_amount);
				printf("Items Per City:				%f\n", problem.items_per_city);
			}
			// Extract the knapsack capacity
			else if (strncmp(str, KNAPSACK_CAPACITY, strlen(KNAPSACK_CAPACITY)) == 0)
			{
				subString(str, sub, position + 1, strlen(str) - position);
				problem.knapsack_capacity = atof(sub);
				printf("Knapsack Capacity:			%lf\n", problem.knapsack_capacity);
			}
			// Extract the minimal speed
			else if (strncmp(str, MIN_SPEED, strlen(MIN_SPEED)) == 0)
			{
				subString(str, sub, position + 1, strlen(str) - position);
				problem.min_speed = atof(sub);
				printf("Minimum Speed:				%lf\n", problem.min_speed);
			}
			// Extract the maximum speed
			else if (strncmp(str, MAX_SPEED, strlen(MAX_SPEED)) == 0)
			{
				subString(str, sub, position + 1, strlen(str) - position);
				problem.max_speed = atof(sub);
				printf("Maximum Speed:				%lf\n", problem.max_speed);
			}
			// Extract the renting ratio
			else if (strncmp(str, RENTING_RATIO, strlen(RENTING_RATIO)) == 0)
			{
				subString(str, sub, position + 1, strlen(str) - position);
				problem.renting_ratio = atof(sub);
				printf("Renting Ratio:				%lf\n", problem.renting_ratio);
			}
			// Extract the edge weight type
			else if (strncmp(str, EDGE_WEIGHT_TYPE, strlen(EDGE_WEIGHT_TYPE)) == 0)
			{
				subString(str, sub, position + 1, strlen(str) - position);
				strcpy(edge_weight_type, sub);
				printf("Edge Weight Type is			%s", edge_weight_type);
			}
		}

		// Close file
		fclose(file_open);
		printf("****************************************************************************************\n");
		printf("\n");

#pragma endregion

#pragma region PROGRAM SETUP

		/****************************************************************************************************
		* PRINT CUDA AND GENETIC VALUES
		****************************************************************************************************/
		printf("****************************************************************************************\n");
		printf("PRGRAM SETUP\n");
		printf("****************************************************************************************\n");
		printf("LOCAL SEARCH PROPABILITY:		%f\n", LOCAL_SEARCH_PROBABILITY);
		printf("AMOUNT OF PARENTS:			%d\n", SELECTED_PARENTS);
		printf("TOURNAMENT SIZE:			%d\n", TOURNAMENT_SIZE);
		printf("AMOUNT OF EVOLUTIONS:			%d\n", NUM_EVOLUTIONS);
		printf("AMOUNT OF SOLUTIONS PER EXECUTION:	%d\n", TOURS);
		printf("AMOUNT OF EXECUTIONS:			%d\n", NUMBER_EXECUTIONS);
		printf("****************************************************************************************\n");

#pragma endregion

#pragma region PRINT SIZE OF STRUCT AND VARIABLES
		/****************************************************************************************************
		* PRINT SIZE OF STRUCTS
		****************************************************************************************************/
		printf("****************************************************************************************\n");
		printf("SIZE OF STRUCTS AND VARIABLES\n");
		printf("****************************************************************************************\n");
		printf("ITEM:				%lld\n", sizeof(item));
		printf("NODE:				%lld\n", sizeof(node));
		printf("TOUR:				%lld\n", sizeof(tour));
		printf("POPULATION:			%lld\n", sizeof(population));
		printf("PARAMETERS:			%lld\n", sizeof(parameters));
		printf("****************************************************************************************\n");
#pragma endregion

#pragma region READ FILE VALUES AND DISPLAY DATA

		/*************************************************************************************************
		* POPULATION INITIALIZATION ON HOST (CPU)
		*************************************************************************************************/

		// Obtain the items
		// Calculate amount of rows
		unsigned int item_rows = countMatrixRows(lines[i], ITEMS_SECTION);

		// Calculate amount of columns
		unsigned int item_columns = 4;

		// Validate file consistency
		if (item_rows != problem.items_amount)
		{
			perror("The file information is not consistent. Number of items Inconsistency.\n");
			exit(EXIT_FAILURE);
		}

		// Get matrix
		matrix = extractMatrixFromFile(lines[i], ITEMS_SECTION, problem.items_amount, item_columns);

		// Allocate memory for the array of structs
		item* cpu_item = (item*)malloc(problem.items_amount * sizeof(item));
		if (cpu_item == NULL) {
			fprintf(stderr, "Out of Memory");
			exit(0);
		}

		// Convert to array of struct
		extractItems(matrix, problem.items_amount, cpu_item);

		// Visualize values for item matrix	
		displayItems(cpu_item, problem.items_amount);

		// Obtain nodes
		// Calculate amount of nodes
		unsigned int node_rows = countMatrixRows(lines[i], NODE_COORD_SECTION);

		// Validate file consistency
		if (node_rows != problem.cities_amount)
		{
			perror("The file information is not consistent. Number of node Inconsistency.\n");
			exit(EXIT_FAILURE);
		}

		// Calculate amount of columns
		unsigned int node_columns = 3;

		// Get matrix
		matrix = extractMatrixFromFile(lines[i], NODE_COORD_SECTION, problem.cities_amount, node_columns);

		// Allocate memory for the array of structs
		node* cpu_node = (node*)malloc(problem.cities_amount * sizeof(node));
		if (cpu_node == NULL) {
			fprintf(stderr, "Out of Memory");
			exit(0);
		}
		// Convert to array of struct
		extractNodes(matrix, problem.cities_amount, cpu_node);

		// Assign items to node
		assignItems(cpu_item, cpu_node);

		// Print node information
		displayNodes(cpu_node, problem.cities_amount);

		// Create Global Stats File 
		createGlobalStatsFile(fname, GPU, CPU);

		// Create Global Output File
		createGlobalOutputFile(fname, GPU, CPU);

#pragma endregion

		for (int clockCounter = 0; clockCounter < NUMBER_EXECUTIONS; ++clockCounter)
		{
#pragma region CENTRAL PROCESSING UNIT
			if (CPU)
			{
				// Create File For Statistics
				if(WRITE_STATS_PER_METHOD)
					createStatisticsFile(fname, false, CPU, clockCounter);

				// Create Results File 
				if(WRITE_RESULTS_PER_ITERATION)
					createOutputFile(fname, false, CPU, clockCounter);

				cpu_start = std::chrono::high_resolution_clock::now();

				if (timespec_get(&startCPU, TIME_UTC) != TIME_UTC)
				{
					printf("Error in calling timespec_get\n");
					exit(EXIT_FAILURE);
				}

				// Assign nodes to tour		
				defineInitialTour(initial_tour, &problem, cpu_node, cpu_item);

				if(WRITE_STATS_PER_METHOD)
				{
					cpu_method_start = std::chrono::high_resolution_clock::now();
				}

				// Initialize population by generating POPULATION_SIZE number of permutations of the initial tour, all starting at the same city
				initializePopulation(initial_population_cpu, initial_tour, problem);

				if (WRITE_STATS_PER_METHOD)
				{
					cpu_method_stop = std::chrono::high_resolution_clock::now();
					elapsedTimeInitialPopulationCPU = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_method_stop - cpu_method_start).count();
					saveStatistics(fname, NO_CUDA, 0, clockCounter, elapsedTimeInitialPopulationCPU, -1, -1, -1);
				}

				if (WRITE_RESULTS_PER_ITERATION)
				{
					fittestOnEarth = getFittestTour(initial_population_cpu.tours, TOURS);
					saveFittest(fname, fittestOnEarth, problem, 0, NO_CUDA, clockCounter);
				}

				if (TIME_RESTRICTED)
				{
					time_t endwait;
					time_t start = time(NULL);

					endwait = start + EXECUTION_TIME;

					int iterationCount = 0;

					while (start < endwait)
					{
						// Select the best parents of the current generation
						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_start = std::chrono::high_resolution_clock::now();
						}
						selection(initial_population_cpu, host_parents);
						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_stop = std::chrono::high_resolution_clock::now();
							elapsedSelectionCPU = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_method_stop - cpu_method_start).count();
						}

						// Breed the population performing crossover (Combination of Ordered Crossover 
						// for the TSP sub-problem and One Point Crossover for the KP sub-problem)
						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_start = std::chrono::high_resolution_clock::now();
						}
						crossover(initial_population_cpu, host_parents, problem);
						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_stop = std::chrono::high_resolution_clock::now();
							elapsedCrossoverCPU = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_method_stop - cpu_method_start).count();
						}

						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_start = std::chrono::high_resolution_clock::now();
						}
						localSearch(initial_population_cpu, problem);
						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_stop = std::chrono::high_resolution_clock::now();
							elapsedLocalSearchCPU = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_method_stop - cpu_method_start).count();
						}

						// Get Fittest tour of the generation
						fittestOnEarth = getFittestTour(initial_population_cpu.tours, TOURS);
						if (WRITE_RESULTS_PER_ITERATION)
						{
							saveFittest(fname, fittestOnEarth, problem, iterationCount + 1, NO_CUDA, clockCounter);
						}
						
						if (WRITE_STATS_PER_METHOD)
						{
							saveStatistics(fname, NO_CUDA, clockCounter, iterationCount + 1, elapsedTimeInitialPopulationCPU, elapsedSelectionCPU, elapsedCrossoverCPU, elapsedLocalSearchCPU);
						}

						Sleep(1);
						start = time(NULL);
					}

					saveGlobalFittest(fname, fittestOnEarth, problem, clockCounter, NO_CUDA);
				}
				else
				{
					for (int i = 0; i < NUM_EVOLUTIONS; ++i)
					{
						// Select the best parents of the current generation
						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_start = std::chrono::high_resolution_clock::now();
						}
						selection(initial_population_cpu, host_parents);
						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_stop = std::chrono::high_resolution_clock::now();
							elapsedSelectionCPU = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_method_stop - cpu_method_start).count();
						}

						// Breed the population performing crossover (Combination of Ordered Crossover 
						// for the TSP sub-problem and One Point Crossover for the KP sub-problem)
						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_start = std::chrono::high_resolution_clock::now();
						}
						crossover(initial_population_cpu, host_parents, problem);
						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_stop = std::chrono::high_resolution_clock::now();
							elapsedCrossoverCPU = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_method_stop - cpu_method_start).count();
						}

						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_start = std::chrono::high_resolution_clock::now();
						}
						localSearch(initial_population_cpu, problem);
						if (WRITE_STATS_PER_METHOD)
						{
							cpu_method_stop = std::chrono::high_resolution_clock::now();
							elapsedLocalSearchCPU = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_method_stop - cpu_method_start).count();
						}

						// Get Fittest tour of the generation
						fittestOnEarth = getFittestTour(initial_population_cpu.tours, TOURS);

						if (WRITE_RESULTS_PER_ITERATION)
						{
							saveFittest(fname, fittestOnEarth, problem, i + 1, NO_CUDA, clockCounter);
						}

						if (WRITE_STATS_PER_METHOD)
						{
							saveStatistics(fname, NO_CUDA, clockCounter, i + 1, elapsedTimeInitialPopulationCPU, elapsedSelectionCPU, elapsedCrossoverCPU, elapsedLocalSearchCPU);
						}
					}
					saveGlobalFittest(fname, fittestOnEarth, problem, clockCounter, NO_CUDA);
				}

				cpu_stop = std::chrono::high_resolution_clock::now();
				elapsedTimeCPU = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_stop - cpu_start).count();
				saveGlobalStatistics(fname, NO_CUDA, clockCounter, elapsedTimeCPU);
			}
#pragma endregion
#pragma region GRAPHICAL PROCESS UNIT
			if (GPU)
			{
				/****************************************************************************************************
				* PRINT PROPERTIES OF THE CUDA DEVICE (IF ANY)
				****************************************************************************************************/
				deviceErr = hipGetDeviceCount(&deviceCount);
				if (deviceCount > 0 && deviceErr == hipSuccess && GPU)
				{
					printf("****************************************************************************************\n");
					printf("PROPERTIES OF THE GRAPHICAL PROCESSING UNIT\n");
					printf("****************************************************************************************\n");
					for (int i = 0; i < deviceCount; i++)
					{
						checkCudaErrors(hipGetDeviceProperties(&properties, i));
						printf("GPU:					%s\n", properties.name);
						printf("Warp Size:				%d\n", properties.warpSize);
						printf("Total Global Memory:			%zd\n", properties.totalGlobalMem);
						printf("Total Constant Memory:			%zd\n", properties.totalConstMem);
						printf("Shared Memory Per Block:		%zd\n", properties.sharedMemPerBlock);
						printf("Multiprocessor:				%d\n", properties.multiProcessorCount);
						printf("Max Threads Per Multiprocessor:		%d\n", properties.maxThreadsPerMultiProcessor);
						printf("Max Blocks Per Multiprocessor:		%d\n", properties.maxBlocksPerMultiProcessor);
						printf("Max Threads Per Block:			%d\n", properties.maxThreadsPerBlock);
					}
					printf("****************************************************************************************\n");
				}

				// Create File For Statistics
				if (WRITE_STATS_PER_METHOD)
					createStatisticsFile(fname, GPU, false, clockCounter);

				// Create Results File 
				if (WRITE_RESULTS_PER_ITERATION)
					createOutputFile(fname, GPU, false, clockCounter);

				cpu_start = std::chrono::high_resolution_clock::now();

				// Assign nodes to tour		
				defineInitialTour(initial_tour, &problem, cpu_node, cpu_item);

				cpu_stop = std::chrono::high_resolution_clock::now();

				elapsedTimeGPU = std::chrono::duration_cast<std::chrono::milliseconds>(cpu_stop - cpu_start).count();

				if (deviceCount > 0 && deviceErr == hipSuccess && GPU)
				{
					checkCudaErrors(hipEventCreate(&startGPU));
					checkCudaErrors(hipEventCreate(&stopGPU));
					checkCudaErrors(hipEventRecord(startGPU, 0));
					/*************************************************************************************************
					* ALLOCATE MEMORY FOR STRUCTS ON DEVICE
					*************************************************************************************************/

					// Allocate device memory for population
					checkCudaErrors(hipMalloc((void**)&device_population, sizeof(population) * size_t(population_size)));

					// Allocate device memory for initial tour
					checkCudaErrors(hipMalloc((void**)&device_initial_tour, sizeof(tour)));

					// Allocate device memory for parents selected from tournament selection
					checkCudaErrors(hipMalloc((void**)&device_parents, sizeof(tour) * size_t(tour_size) * SELECTED_PARENTS));

					// Allocate device memory for states
					checkCudaErrors(hipMalloc((void**)&device_states, sizeof(hiprandState) * TOURS * size_t(problem.cities_amount)));

					// Allocate device memory for the descendants
					checkCudaErrors(hipMalloc((void**)&device_offspring, sizeof(tour) * size_t(tour_size) * TOURS));

					// Allocate device memory for the parameters
					checkCudaErrors(hipMalloc((void**)&device_parameters, sizeof(parameters)));

					// Allocate device memory for node matrix, node matrix transpose and distance matrix
					//checkCudaErrors(hipMalloc(&device_node_matrix, size_t(problem.cities_amount) * sizeof(node)));
					//checkCudaErrors(hipMalloc(&device_node_t_matrix, size_t(problem.cities_amount) * sizeof(node)));
					//checkCudaErrors(hipMalloc(&device_distance, sizeof(distance) * CITIES * CITIES));

					/*************************************************************************************************
					* COPY HOST MEMORY TO DEVICE
					*************************************************************************************************/

					// Copy problem data
					checkCudaErrors(hipMemcpy(device_parameters, &problem, sizeof(parameters), hipMemcpyHostToDevice));

					// Copy initial tour data
					checkCudaErrors(hipMemcpy(device_initial_tour, &initial_tour, sizeof(tour), hipMemcpyHostToDevice));

					// Copy node data
					//checkCudaErrors(hipMemcpy(device_node_matrix, cpu_node, size_t(problem.cities_amount) * sizeof(node), hipMemcpyHostToDevice));

					/*************************************************************************************************
					* INITIALIZE RANDOM VALUES
					*************************************************************************************************/
					initCuRand << <numBlocks, threadsPerBlock >> > (device_states, time(NULL));
					//initCuRand << <BLOCKS, THREADS >> > (device_states, time(NULL));
					checkCudaErrors(hipDeviceSynchronize());

					/*************************************************************************************************
					* POPULATION INITIALIZATION ON DEVICE (GPU)
					*************************************************************************************************/
					if (WRITE_STATS_PER_METHOD)
					{
						checkCudaErrors(hipEventCreate(&startKernel));
						checkCudaErrors(hipEventCreate(&stopKernel));
						checkCudaErrors(hipEventRecord(startKernel, 0));
					}
					initializePopulationCuda << <numBlocks, threadsPerBlock >> > (device_population, device_initial_tour, device_parameters, device_states);
					//initializePopulationCuda << <BLOCKS, THREADS >> > (device_population, device_initial_tour, device_parameters, device_states);
					if (WRITE_STATS_PER_METHOD)
					{
						checkCudaErrors(hipEventRecord(stopKernel, 0));
						checkCudaErrors(hipEventSynchronize(stopKernel));
						checkCudaErrors(hipEventElapsedTime(&elapsedTimeInitialPopulationGPU, startKernel, stopKernel));
						saveStatistics(fname, CUDA, 0, clockCounter, elapsedTimeInitialPopulationGPU, -1, -1, -1);
					}
					//checkCudaErrors(hipDeviceSynchronize());

					/*************************************************************************************************
					* COPY RESULTS TO HOST - OPTIONAL: REMOVE FOR PERFORMANCE
					*************************************************************************************************/
					//checkCudaErrors(hipMemcpy(&initial_population_gpu, device_population, sizeof(population), hipMemcpyDeviceToHost));
					//checkCudaErrors(hipDeviceSynchronize());

					/*************************************************************************************************
					* OUTPUT - OPTIONAL: REMOVE FOR PERFORMANCE
					*************************************************************************************************/
					//printPopulation(initial_population_gpu);
					//saveInitialPopulation(problem.name, initial_population_gpu, problem, CUDA, clockCounter, elapsedTimeInitialPopulationGPU[clockCounter]);

					// Copy Device Information to Host
					//checkCudaErrors(hipMemcpy(&initial_population_gpu, device_population, sizeof(population), hipMemcpyDeviceToHost));
					//checkCudaErrors(hipDeviceSynchronize());

					if (WRITE_RESULTS_PER_ITERATION)
					{
						fittestOnEarth = getFittestTour(initial_population_gpu.tours, TOURS);
						saveFittest(fname, fittestOnEarth, problem, 0, CUDA, clockCounter);
					}

					if (TIME_RESTRICTED)
					{
						time_t endwait;
						time_t start = time(NULL);

						endwait = start + EXECUTION_TIME;

						int iterationCount = 0;

						while (start < endwait)
						{
							// Select Parents For The Next Generation
							if (WRITE_STATS_PER_METHOD)
								checkCudaErrors(hipEventRecord(startKernel, 0));
							selectionKernel << <numBlocks, threadsPerBlock >> > (device_population, device_parents, device_states);
							err = hipGetLastError();
							if (err != hipSuccess)
							{
								fprintf(stderr, "Selection Kernel: %s\n", hipGetErrorString(err));
								exit(0);
							}
							if (WRITE_STATS_PER_METHOD)
							{
								checkCudaErrors(hipEventRecord(stopKernel, 0));
								checkCudaErrors(hipEventSynchronize(stopKernel));
								checkCudaErrors(hipEventElapsedTime(&elapsedSelectionGPU, startKernel, stopKernel));
							}							

							// Breed the population performing crossover (Combination of Ordered Crossover 
							// for the TSP sub-problem and One Point Crossover for the KP sub-problem)
							if (WRITE_STATS_PER_METHOD)
								checkCudaErrors(hipEventRecord(startKernel, 0));
							crossoverKernel << <numBlocks, threadsPerBlock >> > (device_population, device_parents, device_offspring, device_parameters, device_states);
							err = hipGetLastError();
							if (err != hipSuccess)
							{
								fprintf(stderr, "Crossover Kernel: %s\n", hipGetErrorString(err));
								exit(0);
							}
							if (WRITE_STATS_PER_METHOD)
							{
								checkCudaErrors(hipEventRecord(stopKernel, 0));
								checkCudaErrors(hipEventSynchronize(stopKernel));
								checkCudaErrors(hipEventElapsedTime(&elapsedCrossoverGPU, startKernel, stopKernel));
							}

							// Perform local search (mutation)
							if (WRITE_STATS_PER_METHOD)
								checkCudaErrors(hipEventRecord(startKernel, 0));
							localSearchKernel << <numBlocks, threadsPerBlock >> > (device_population, device_parameters, device_states);
							err = hipGetLastError();
							if (err != hipSuccess)
							{
								fprintf(stderr, "Local Search Kernel: %s\n", hipGetErrorString(err));
								exit(0);
							}
							if (WRITE_STATS_PER_METHOD)
							{
								checkCudaErrors(hipEventRecord(stopKernel, 0));
								checkCudaErrors(hipEventSynchronize(stopKernel));
								checkCudaErrors(hipEventElapsedTime(&elapsedLocalSearchGPU, startKernel, stopKernel));
							}

							// Copy Device Information to Host
							//checkCudaErrors(hipMemcpy(&initial_population_gpu, device_population, sizeof(population), hipMemcpyDeviceToHost));
							//checkCudaErrors(hipDeviceSynchronize());

							if (WRITE_RESULTS_PER_ITERATION)
							{
								// Get Fittest tour of the generation
								fittestOnEarth = getFittestTour(initial_population_gpu.tours, TOURS);
								saveFittest(fname, fittestOnEarth, problem, iterationCount + 1, CUDA, clockCounter);
							}

							if (WRITE_STATS_PER_METHOD)
								saveStatistics(fname, CUDA, clockCounter, iterationCount + 1, elapsedTimeInitialPopulationGPU, elapsedSelectionGPU, elapsedCrossoverGPU, elapsedLocalSearchGPU);
							
							iterationCount++;
							Sleep(1);
							start = time(NULL);
						}

						// Copy Device Information to Host
						checkCudaErrors(hipMemcpy(&initial_population_gpu, device_population, sizeof(population), hipMemcpyDeviceToHost));
						checkCudaErrors(hipDeviceSynchronize());

						// Get Fittest tour of the execution
						fittestOnEarth = getFittestTour(initial_population_gpu.tours, TOURS);
						saveGlobalFittest(fname, fittestOnEarth, problem, clockCounter, CUDA);
					}
					else
					{
						for (int i = 0; i < NUM_EVOLUTIONS; ++i)
						{
							// Select Parents For The Next Generation
							if (WRITE_STATS_PER_METHOD)
								checkCudaErrors(hipEventRecord(startKernel, 0));
							selectionKernel << <numBlocks, threadsPerBlock >> > (device_population, device_parents, device_states);
							//selectionKernel << <BLOCKS, THREADS >> > (device_population, device_parents, device_states);
							err = hipGetLastError();
							if (err != hipSuccess)
							{
								fprintf(stderr, "Selection Kernel: %s\n", hipGetErrorString(err));
								exit(0);
							}
							if (WRITE_STATS_PER_METHOD)
							{
								checkCudaErrors(hipEventRecord(stopKernel, 0));
								checkCudaErrors(hipEventSynchronize(stopKernel));
								checkCudaErrors(hipEventElapsedTime(&elapsedSelectionGPU, startKernel, stopKernel));
							}
							//checkCudaErrors(hipDeviceSynchronize());

							// Copy Device Information to Host
							//checkCudaErrors(hipMemcpy(&host_parents, device_parents, sizeof(tour) * SELECTED_PARENTS, hipMemcpyDeviceToHost));
							//checkCudaErrors(hipDeviceSynchronize());

							// Save Parents Information To File
							//saveParents(problem.name, host_parents, problem, i + 1, CUDA, clockCounter, elapsedSelectionGPU[i]);

							// Breed the population performing crossover (Combination of Ordered Crossover 
							// for the TSP sub-problem and One Point Crossover for the KP sub-problem)
							if (WRITE_STATS_PER_METHOD)
								checkCudaErrors(hipEventRecord(startKernel, 0));
							crossoverKernel << <numBlocks, threadsPerBlock >> > (device_population, device_parents, device_offspring, device_parameters, device_states);
							//crossoverKernel << <BLOCKS, THREADS >> > (device_population, device_parents, device_offspring, device_parameters, device_states);
							err = hipGetLastError();
							if (err != hipSuccess)
							{
								fprintf(stderr, "Crossover Kernel: %s\n", hipGetErrorString(err));
								exit(0);
							}
							if (WRITE_STATS_PER_METHOD)
							{
								checkCudaErrors(hipEventRecord(stopKernel, 0));
								checkCudaErrors(hipEventSynchronize(stopKernel));
								checkCudaErrors(hipEventElapsedTime(&elapsedCrossoverGPU, startKernel, stopKernel));
							}
							//checkCudaErrors(hipDeviceSynchronize());

							// Perform local search (mutation)
							if (WRITE_STATS_PER_METHOD)
								checkCudaErrors(hipEventRecord(startKernel, 0));
							localSearchKernel << <numBlocks, threadsPerBlock >> > (device_population, device_parameters, device_states);
							//localSearchKernel << <BLOCKS, THREADS >> > (device_population, device_parameters, device_states);
							err = hipGetLastError();
							if (err != hipSuccess)
							{
								fprintf(stderr, "Local Search Kernel: %s\n", hipGetErrorString(err));
								exit(0);
							}
							if (WRITE_STATS_PER_METHOD)
							{
								checkCudaErrors(hipEventRecord(stopKernel, 0));
								checkCudaErrors(hipEventSynchronize(stopKernel));
								checkCudaErrors(hipEventElapsedTime(&elapsedLocalSearchGPU, startKernel, stopKernel));
							}
							//checkCudaErrors(hipDeviceSynchronize());

							// Copy Device Information to Host
							//checkCudaErrors(hipMemcpy(&initial_population_gpu, device_population, sizeof(population), hipMemcpyDeviceToHost));
							//checkCudaErrors(hipDeviceSynchronize());

							//saveOffspring(problem.name, initial_population_gpu, problem, i + 1, CUDA, clockCounter, elapsedCrossoverGPU[i], elapsedLocalSearchGPU[i]);

							if (WRITE_RESULTS_PER_ITERATION)
							{
								// Get Fittest tour of the generation
								fittestOnEarth = getFittestTour(initial_population_gpu.tours, TOURS);
								saveFittest(fname, fittestOnEarth, problem, i + 1, CUDA, clockCounter);
							}

							if (WRITE_STATS_PER_METHOD)
								saveStatistics(fname, CUDA, clockCounter, i + 1, elapsedTimeInitialPopulationGPU, elapsedSelectionGPU, elapsedCrossoverGPU, elapsedLocalSearchGPU);
						}

						// Copy Device Information to Host
						checkCudaErrors(hipMemcpy(&initial_population_gpu, device_population, sizeof(population), hipMemcpyDeviceToHost));
						checkCudaErrors(hipDeviceSynchronize());

						// Get Fittest tour of the execution
						fittestOnEarth = getFittestTour(initial_population_gpu.tours, TOURS);
						saveGlobalFittest(fname, fittestOnEarth, problem, clockCounter, CUDA);
					}

					checkCudaErrors(hipEventRecord(stopGPU, 0));
					checkCudaErrors(hipEventSynchronize(stopGPU));
					checkCudaErrors(hipEventElapsedTime(&gpuExecutionTime, startGPU, stopGPU));
					elapsedTimeGPU += gpuExecutionTime;

					/*************************************************************************************************
					* RELEASE CUDA MEMORY
					*************************************************************************************************/
					checkCudaErrors(hipFree(device_population));
					checkCudaErrors(hipFree(device_initial_tour));
					checkCudaErrors(hipFree(device_parents));
					checkCudaErrors(hipFree(device_offspring));
					checkCudaErrors(hipFree(device_parameters));
					checkCudaErrors(hipFree(device_states));
					//hipFree(device_node_matrix);
					//hipFree(device_node_t_matrix);
					//hipFree(device_distance);
					checkCudaErrors(hipEventDestroy(startGPU));
					checkCudaErrors(hipEventDestroy(startKernel));
					checkCudaErrors(hipEventDestroy(stopKernel));
					checkCudaErrors(hipEventDestroy(stopGPU));
				}

				saveGlobalStatistics(fname, CUDA, clockCounter, elapsedTimeGPU);
			}
#pragma endregion

			// Calculate distance matrix in CPU
			//int distance_matrix_size = problem.cities_amount * problem.cities_amount;

			// Allocate memory for the distance matrix
			//distance* d = (distance*)malloc(distance_matrix_size * sizeof(distance));
			//if (d == NULL) {
			//	fprintf(stderr, "Out of Memory");
			//	exit(0);
			//}

			//euclideanDistanceCPU(cpu_node, cpu_node, d, problem.cities_amount, distance_matrix_size);
			//displayDistance(d, distance_matrix_size);

#pragma region DISTANCE MATRIX GRAPHICAL PROCESSING UNIT

		//if (deviceCount > 0 && deviceErr == hipSuccess && GPU)
		//{
			/*************************************************************************************************
			* CALCULATE DISTANCE MATRIX IN CUDA
			*************************************************************************************************/

			// Execute CUDA Matrix Transposition
			//printf("Transponiendo la matrix de nodos de tamaño [%d][%d]\n", node_rows, 1);
			//transpose << <BLOCKS, THREADS >> > (device_node_matrix, device_node_t_matrix, node_rows, 1);
			//checkCudaErrors(hipDeviceSynchronize());

			// Copy results from device to host
			//node* h_node_t_matrix = (node*)malloc(sizeof(node) * problem.cities_amount);
			//checkCudaErrors(hipMemcpy(h_node_t_matrix, device_node_t_matrix, sizeof(node) * problem.cities_amount, hipMemcpyDeviceToHost));

			// Show information on screen
			//displayNodes(h_node_t_matrix, problem.cities_amount);

			// TODO: FIX GRID AND THREADS AND MATRIXDISTANCES KERNEL
			//dim3 grid(8, 8, 1);
			//dim3 threads(BLOCK_SIZE, BLOCK_SIZE, 1);

			//printf("Calculando la matriz de distancias en GPU\n");
			//matrixDistances << <grid, threads >> > (device_node_matrix, device_node_t_matrix, device_distance, problem.cities_amount, problem.cities_amount);
			//checkCudaErrors(hipDeviceSynchronize());

			//Copy results from device to host
			//distance* h_distance = (distance*)malloc(sizeof(distance) * CITIES * CITIES);
			//checkCudaErrors(hipMemcpy(h_distance, device_distance, sizeof(distance) * CITIES * CITIES, hipMemcpyDeviceToHost));

			// Show Data
			//displayDistance(h_distance, CITIES * CITIES);
		//}
#pragma endregion		
		}

		free(cpu_item);
		free(cpu_node);
	}
	return 0;
}